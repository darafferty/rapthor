#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>

#include "Types.h"
#include "math.cu"

extern "C" {

/*
	Kernel
*/
__global__ void kernel_gridder(
	const int bl_offset,
	const UVWType			__restrict__ uvw,
	const WavenumberType	__restrict__ wavenumbers,
	const VisibilitiesType	__restrict__ visibilities,
	const SpheroidalType	__restrict__ spheroidal,
	const ATermType			__restrict__ aterm,
	const BaselineType		__restrict__ baselines,		
	SubGridType				__restrict__ subgrid
	) {
	int tidx = threadIdx.x;
	int tidy = threadIdx.y;
	int tid = tidx + tidy * blockDim.x;
	int bl = blockIdx.x + blockIdx.y * gridDim.x;

    // Shared data
	__shared__ UVW _uvw[CHUNKSIZE];
	__shared__ float _wavenumbers[NR_CHANNELS];
	__shared__ float2 _visibilities[NR_CHANNELS][NR_POLARIZATIONS];
	
    // Load wavenumbers
    if (tid < NR_CHANNELS) {
        _wavenumbers[tid] = wavenumbers[tid];
    }
    __syncthreads();
	
	for (int chunk = 0; chunk < NR_CHUNKS; chunk++) {
        // Load UVW	
	    if (tid < CHUNKSIZE) {
	        _uvw[tid] = uvw[bl][tid];	    
	    }
	    __syncthreads();
	
	    // Compute offset for current chunk
	    UVW uvw_first = _uvw[0];
        UVW uvw_last = _uvw[CHUNKSIZE-1];
	    int u = (GRIDSIZE/2) - ((uvw_first.u + uvw_last.u) / 2);
	    int v = (GRIDSIZE/2) - ((uvw_first.v + uvw_last.v) / 2);
	    int w = (GRIDSIZE/2) - ((uvw_first.w + uvw_last.w) / 2);
	    UVW _offset = {u, v, w};
	    
        // Iterate all pixels in subgrid
        for (int y = tidy; y < SUBGRIDSIZE; y += blockDim.y) {
	        for (int x = tidx; x < SUBGRIDSIZE; x += blockDim.x) {
	            // Private subgrid points
	            float2 uvXX = {0, 0};
	            float2 uvXY = {0, 0};
	            float2 uvYX = {0, 0};
	            float2 uvYY = {0, 0};
	        
	            // Compute l,m,n
	            float l = -(x-(SUBGRIDSIZE/2)) * IMAGESIZE/SUBGRIDSIZE;
	            float m =  (y-(SUBGRIDSIZE/2)) * IMAGESIZE/SUBGRIDSIZE;
                float n = 1.0f - (float) sqrt(1.0 - (double) (l * l) - (double) (m * m));

                // Iterate all timesteps in current chunk
	            for (int time = 0; time < CHUNKSIZE && (chunk * CHUNKSIZE) < NR_TIME; time++) {
            	    // Load visibilities for all channels and polarizations
                	for (int i = tid; i < NR_CHANNELS * NR_POLARIZATIONS; i += blockDim.x * blockDim.y) {
                	    _visibilities[0][i] = visibilities[bl][time][0][i];
                	}
	            
	                 // Load UVW coordinates
		            float u = _uvw[time].u;
		            float v = _uvw[time].v;
		            float w = _uvw[time].w;
		
		            // Compute phase index
                	float ulvmwn = u*l + v*m + w*n;

		            // Compute phase offset
		            float phase_offset = _offset.u*l +
						                 _offset.v*m +
						                 _offset.w*n;
						                 
		            // Compute phasor
		            float phasor_real[NR_CHANNELS];
		            float phasor_imag[NR_CHANNELS];
		            for (int chan = 0; chan < NR_CHANNELS; chan++) {
		                float phase = (ulvmwn * _wavenumbers[chan]) - phase_offset;
                        float2 phasor = make_float2(0, 0);
                        phasor_real[chan] = cos(phase);
                        phasor_imag[chan] = sin(phase);
                    }

		            // Sum updates for all channels
		            for (int chan = 0; chan < NR_CHANNELS; chan++) {
			            // Load visibilities from shared memory
			            float2 visXX = _visibilities[chan][0];
			            float2 visXY = _visibilities[chan][1];
			            float2 visYX = _visibilities[chan][2];
			            float2 visYY = _visibilities[chan][3];
			                	
			            // Load phasor
                        float2 phasor = make_float2(phasor_real[chan], phasor_imag[chan]);
			
			            // Multiply visibility by phasor
			            uvXX += phasor * visXX;
			            uvXY += phasor * visXY;
			            uvYX += phasor * visYX;
			            uvYY += phasor * visYY;
		            }
	            }
	
	            // Load spheroidal
	            float s = spheroidal[y][x];
	
	            // Stations for current baseline
                int station1 = baselines[bl+bl_offset].station1;
	            int station2 = baselines[bl+bl_offset].station2;
	
                // Get a term for station1
	            float2 aXX1 = hipConjf(aterm[station1][0][y][x]);
	            float2 aXY1 = hipConjf(aterm[station1][1][y][x]);
	            float2 aYX1 = hipConjf(aterm[station1][2][y][x]);
	            float2 aYY1 = hipConjf(aterm[station1][3][y][x]);

	            // Get aterm for station2
	            float2 aXX2 = aterm[station2][0][y][x];
	            float2 aXY2 = aterm[station2][1][y][x];
	            float2 aYX2 = aterm[station2][2][y][x];
	            float2 aYY2 = aterm[station2][3][y][x];
	
	            // Apply aterm
	            float2 _uvXX = uvXX;
	            float2 _uvXY = uvXY;
	            float2 _uvYX = uvYX;
	            float2 _uvYY = uvYY;
	            uvXX  = _uvXX * aXX1;
	            uvXX += _uvXY * aYX1; 
	            uvXX += _uvXX * aXX2;
	            uvXX += _uvXY * aYX2;
	            uvXY  = _uvXX * aXY1;
	            uvXY += _uvXY * aYY1;
	            uvXY += _uvXX * aXY2;
	            uvXY += _uvXY * aYY2;
	            uvYX  = _uvYX * aXX1;
	            uvYX += _uvYY * aYX1;
	            uvYX += _uvYX * aXX2;
	            uvYX += _uvYY * aYX2;
	            uvYY  = _uvYY * aXY1;
	            uvYY += _uvYY * aYY1;
	            uvYY += _uvYY * aXY2;
	            uvYY += _uvYY * aYY2;
	
	            // Apply spheroidal and update uv grid
                #if ORDER == ORDER_BL_P_V_U
	            subgrid[bl][chunk][0][y][x] = uvXX * s;
	            subgrid[bl][chunk][1][y][x] = uvXY * s;
	            subgrid[bl][chunk][2][y][x] = uvYX * s;
	            subgrid[bl][chunk][3][y][x] = uvYY * s;
	            #elif ORDER_BL_V_U_P
	            subgrid[bl][chunk][y][x][0] = uvXX * s;
	            subgrid[bl][chunk][y][x][1] = uvXY * s;
	            subgrid[bl][chunk][y][x][2] = uvYX * s;
	            subgrid[bl][chunk][y][x][3] = uvYY * s;
	            #endif
            }
        }
    }
}
}
