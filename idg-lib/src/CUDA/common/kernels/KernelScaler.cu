#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>

#include "Types.h"
#include "math.cu"

extern "C" {

/*
    Kernel
*/
__global__ void kernel_scaler(
    int subgrid_size,
    float2* __restrict__ subgrid
    )
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int tid = tidx + tidy * blockDim.x;
    int nr_threads = blockDim.x * blockDim.y;
    int s = blockIdx.x;

    // Compute scaling factor
    float scale = 1 / (float(subgrid_size)*float(subgrid_size));

    // Iterate all pixels in subgrid
    for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads) {
        int y = i / subgrid_size;
        int x = i % subgrid_size;

        if (y < subgrid_size) {
            for (int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                int idx = index_subgrid(subgrid_size, s, pol, y, x);
                subgrid[idx] = subgrid[idx] * scale;
            }
        }
    }
}
}
