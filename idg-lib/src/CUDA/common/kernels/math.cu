#include "hip/hip_runtime.h"
// Copyright (C) 2020 ASTRON (Netherlands Institute for Radio Astronomy)
// SPDX-License-Identifier: GPL-3.0-or-later

#include <hip/hip_complex.h>
inline __device__ float2 conj(float2 a) {
    return hipConjf(a);
}

inline __device__ float2 operator+(float2 a, float2 b) {
    return make_float2(a.x + b.x, a.y + b.y);
}

inline __device__ float2 operator-(float2 a, float2 b) {
    return make_float2(a.x - b.x, a.y - b.y);
}

inline __device__ float2 operator*(float2 a, float b) {
    return make_float2(a.x * b, a.y * b);
}

inline __device__ float2 operator*(float a, float2 b) {
    return make_float2(a * b.x, a * b.y);
}

inline __device__ float2 operator*(float2 a, float2 b) {
    return make_float2(a.x * b.x - a.y * b.y,
                       a.x * b.y + a.y * b.x);
}

inline __device__ float4 operator*(float4 a, float b) {
    return make_float4(a.x * b, a.y * b, a.z * b, a.w * b);
}

inline __device__ float4 operator*(float a, float4 b) {
    return make_float4(a * b.x, a * b.y, a * b.z, a * b.w);
}

inline __device__ void operator+=(float2 &a, float2 b) {
    a.x += b.x;
    a.y += b.y;
}

inline __device__ void operator+=(double2 &a, double2 b) {
    a.x += b.x;
    a.y += b.y;
}

inline __device__ void operator+=(float4 &a, float4 b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
    a.w += b.w;
}

#if __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(
			address_as_ull, assumed,
			__double_as_longlong(val + __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

inline  __device__ void atomicAdd(float2 &a, float2 b) {
    atomicAdd(&(a.x), b.x);
    atomicAdd(&(a.y), b.y);
}

inline  __device__ void atomicAdd(double2 &a, double2 b) {
    atomicAdd(&(a.x), b.x);
    atomicAdd(&(a.y), b.y);
}

inline __device__ float raw_sin(float a)
{
    float r;
    asm ("sin.approx.ftz.f32 %0,%1;" : "=f"(r) : "f"(a));
    return r;
}

inline __device__ float raw_cos(float a)
{
    float r;
    asm ("cos.approx.ftz.f32 %0,%1;" : "=f"(r) : "f"(a));
    return r;
}


/*
    Multiply accumulate: a = a + (b * c)
*/
// scalar
inline __device__ void mac(float &a, float b, float c)
{
    asm ("fma.rn.ftz.f32 %0,%1,%2,%3;" : "=f"(a) : "f"(b), "f"(c), "f"(a));
}

// complex
inline __device__ void cmac(float2 &a, float2 b, float2 c)
{
    asm ("fma.rn.ftz.f32 %0,%1,%2,%3;" : "=f"(a.x) : "f"(b.x), "f"(c.x), "f"(a.x));
    asm ("fma.rn.ftz.f32 %0,%1,%2,%3;" : "=f"(a.y) : "f"(b.x), "f"(c.y), "f"(a.y));
    asm ("fma.rn.ftz.f32 %0,%1,%2,%3;" : "=f"(a.x) : "f"(-b.y), "f"(c.y), "f"(a.x));
    asm ("fma.rn.ftz.f32 %0,%1,%2,%3;" : "=f"(a.y) : "f"(b.y), "f"(c.x), "f"(a.y));
}

template <typename T>
inline __device__ void apply_avg_aterm_correction_(
    const T C[16], T pixels[4]) {

  const T p[4] = {pixels[0], pixels[2], pixels[1], pixels[3]};

  #pragma unroll 1
  for (int i = 0; i < 4; i++)
  {
    int offset = 0;
    switch (i) {
        case 1: offset = 8; break;
        case 2: offset = 4; break;
        case 3: offset = 12; break;
    }
    pixels[i]  = p[0] * C[offset + 0];
    pixels[i] += p[1] * C[offset + 1];
    pixels[i] += p[2] * C[offset + 2];
    pixels[i] += p[3] * C[offset + 3];
  }
}

/*
    Common math functions
*/
#define FUNCTION_ATTRIBUTES __device__
#include "common/Math.h"
