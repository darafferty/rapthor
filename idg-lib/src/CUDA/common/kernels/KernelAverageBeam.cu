#include "hip/hip_runtime.h"
// Copyright (C) 2020 ASTRON (Netherlands Institute for Radio Astronomy)
// SPDX-License-Identifier: GPL-3.0-or-later

#include "math.cu"
#include "Types.h"

#define BATCH_SIZE 64

inline __device__ size_t index_weight(
    unsigned int nr_time,
    unsigned int nr_channels,
    unsigned int bl,
    unsigned int time,
    unsigned int chan,
    unsigned int pol) {
  // weights: [nr_baselines][nr_time][nr_channels][4]
  return bl * nr_time * nr_channels * 4 +
                 time * nr_channels * 4 +
                               chan * 4 +
                                      pol;
}

inline __device__ size_t index_average_beam(
    unsigned int i,
    unsigned int ii,
    unsigned int jj) {
  // average_beam: [subgrid_size*subgrid_size][4][4]
  return i * 4 * 4 +
            ii * 4 +
                  jj;
}

/*
    Kernel
*/
extern "C" {
__global__ void kernel_average_beam(
    const unsigned int                  nr_antennas,
    const unsigned int                  nr_timesteps,
    const unsigned int                  nr_channels,
    const unsigned int                  nr_aterms,
    const unsigned int                  subgrid_size,
    const UVW<float>*      __restrict__ uvw,
    const Baseline*        __restrict__ baselines,
    const float2*          __restrict__ aterms,
    const int*             __restrict__ aterms_offsets,
    const float*           __restrict__ weights,
         double2*          __restrict__ average_beam)
{
  unsigned int bl = blockIdx.x;
  unsigned int tid = threadIdx.x;
  unsigned int num_threads = blockDim.x;

  unsigned int antenna1 = baselines[bl].station1;
  unsigned int antenna2 = baselines[bl].station2;

  // Check whether stationPair is initialized
  if (antenna1 >= nr_antennas || antenna2 >= nr_antennas) {
    return;
  }

  // Iterate aterms
  for (int aterms_offset = 0; aterms_offset < nr_aterms; aterms_offset += BATCH_SIZE) {
    int current_nr_aterms = min(int(nr_aterms - aterms_offset), BATCH_SIZE);

    float sum_of_weights[BATCH_SIZE][4];
    memset(sum_of_weights, 0, BATCH_SIZE * 4 * sizeof(float));

    // Compute average beam for all pixels
    for (unsigned int i = tid; i < (subgrid_size * subgrid_size); i += num_threads) {
      unsigned y = i / subgrid_size;
      unsigned x = i % subgrid_size;

      double2 sum[4][4] = {0, 0};

      // Loop over aterms
      for (unsigned int n = 0; n < current_nr_aterms; n++) {
        unsigned int aterms_idx = aterms_offset + n;

        // Compute sum of weights
        if (i == tid) {
          unsigned int time_start = aterms_offsets[aterms_idx];
          unsigned int time_end   = aterms_offsets[aterms_idx + 1];

          for (unsigned int t = time_start; t < time_end; t++) {
            float u = uvw[bl * nr_timesteps + t].u;
            if (isinf(u)) continue;

            for (unsigned int ch = 0; ch < nr_channels; ch++) {
              for (unsigned int pol = 0; pol < 4; pol++) {
                  unsigned int weight_idx = index_weight(nr_timesteps, nr_channels, bl, t, ch, pol);
                  sum_of_weights[n][pol] += weights[weight_idx];
              }
            }
          } // end for time
        }

        int station1_idx = index_aterm(subgrid_size, 4, nr_antennas, aterms_idx, antenna1, y, x, 0);
        int station2_idx = index_aterm(subgrid_size, 4, nr_antennas, aterms_idx, antenna2, y, x, 0);

        float2 aXX1 = aterms[station1_idx + 0];
        float2 aXY1 = aterms[station1_idx + 1];
        float2 aYX1 = aterms[station1_idx + 2];
        float2 aYY1 = aterms[station1_idx + 3];
        float2 aXX2 = conj(aterms[station2_idx + 0]);
        float2 aXY2 = conj(aterms[station2_idx + 1]);
        float2 aYX2 = conj(aterms[station2_idx + 2]);
        float2 aYY2 = conj(aterms[station2_idx + 3]);

        float2 kp[16] = {};
        kp[0 + 0] = aXX2 * aXX1;
        kp[0 + 4] = aXX2 * aXY1;
        kp[0 + 8] = aXY2 * aXX1;
        kp[0 + 12] = aXY2 * aXY1;

        kp[1 + 0] = aXX2 * aYX1;
        kp[1 + 4] = aXX2 * aYY1;
        kp[1 + 8] = aXY2 * aYX1;
        kp[1 + 12] = aXY2 * aYY1;

        kp[2 + 0] = aYX2 * aXX1;
        kp[2 + 4] = aYX2 * aXY1;
        kp[2 + 8] = aYY2 * aXX1;
        kp[2 + 12] = aYY2 * aXY1;

        kp[3 + 0] = aYX2 * aYX1;
        kp[3 + 4] = aYX2 * aYY1;
        kp[3 + 8] = aYY2 * aYX1;
        kp[3 + 12] = aYY2 * aYY1;

        for (int ii = 0; ii < 4; ii++) {
          for (int jj = 0; jj < 4; jj++) {
            // Compute real and imaginary part of update separately
            float update_real = 0;
            float update_imag = 0;
            for (int p = 0; p < 4; p++) {
              float weight = sum_of_weights[n][p];
              float kp1_real = kp[4 * ii + p].x;
              float kp1_imag = -kp[4 * ii + p].y;
              float kp2_real = kp[4 * jj + p].x;
              float kp2_imag = kp[4 * jj + p].y;
              update_real +=
                  weight * (kp1_real * kp2_real - kp1_imag * kp2_imag);
              update_imag +=
                  weight * (kp1_real * kp2_imag + kp1_imag * kp2_real);
            }

            // Add kronecker product to sum
            sum[ii][jj] += make_double2(update_real, update_imag);
          }
        }
      } // end for aterms

      // Set average beam from sum of kronecker products
      for (int ii = 0; ii < 4; ii++) {
        for (int jj = 0; jj < 4; jj++) {
          unsigned average_beam_idx = index_average_beam(i, ii, jj);
          atomicAdd(average_beam[average_beam_idx], sum[ii][jj]);
        }
      }
    } // end for pixels
  } // end for aterm_block_idx
}

} // end extern "C"