#include "hip/hip_runtime.h"
#include "Types.h"
#include "math.cu"

#define ALIGN(N,A) (((N)+(A)-1)/(A)*(A))
#define MAX_NR_TERMS     8
#define MAX_SUBGRID_SIZE 32
#define MAX_NR_THREADS   128
#define MAX_NR_TIMESTEPS 128

inline __device__ long index_sum_deriv(
    unsigned int total_nr_timesteps, // number of timesteps for all baselines
    unsigned int nr_channels,        // number channels for a single baseline
    unsigned int term_nr,
    unsigned int pol,
    unsigned int time,
    unsigned int chan)
{
    // sums: [MAX_NR_TERMS][NR_POLARIZATIONS][TOTAL_NR_TIMESTEPS][NR_CHANNELS]
    return term_nr * NR_POLARIZATIONS * total_nr_timesteps * nr_channels +
           pol * total_nr_timesteps * nr_channels +
           time * nr_channels +
           chan;
}

__device__ void compute_lmnp(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const Metadata*      __restrict__ metadata,
          float4                      lmnp_[MAX_SUBGRID_SIZE][MAX_SUBGRID_SIZE])
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned nr_threads = blockDim.x * blockDim.y;
    unsigned s          = blockIdx.x;

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const int x_coordinate      = m.coordinate.x;
    const int y_coordinate      = m.coordinate.y;
    const int z_coordinate      = m.coordinate.z;

    // Compute u,v,w offset in wavelenghts
    const float u_offset = (x_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
    const float v_offset = (y_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
    const float w_offset = w_step * ((float) z_coordinate + 0.5) * 2 * M_PI;

    for (unsigned int i = tid; i < (subgrid_size * subgrid_size); i += nr_threads) {
        unsigned int y = i / subgrid_size;
        unsigned int x = i % subgrid_size;

        if (y < subgrid_size) {
            float l = compute_l(x, subgrid_size, image_size);
            float m = compute_m(y, subgrid_size, image_size);
            float n = compute_n(l, m);
            float phase_offset = u_offset*l + v_offset*m + w_offset*n;
            lmnp_[y][x] = make_float4(l, m, n, phase_offset);
        }
    }
} // end compute_lmnp


template<int current_nr_terms>
__device__ void update_sums(
    const int                         subgrid_size,
    const float                       image_size,
    const unsigned int                total_nr_timesteps,
    const unsigned int                nr_channels,
    const unsigned int                nr_stations,
    const unsigned int                nr_terms,
    const unsigned int                term_offset,
    const UVW*           __restrict__ uvw,
    const float2*        __restrict__ aterm,
    const int*           __restrict__ aterm_indices,
    const float2*        __restrict__ aterm_derivatives,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sum_aterm,
          float2*        __restrict__ sum_deriv,
          float4                      lmnp_[MAX_SUBGRID_SIZE][MAX_SUBGRID_SIZE],
          float2                      pixels_[MAX_NR_TERMS][NR_POLARIZATIONS][MAX_SUBGRID_SIZE])
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned nr_threads = blockDim.x * blockDim.y;
    unsigned s          = blockIdx.x;
    unsigned nr_pixels  = subgrid_size * subgrid_size;

    // Load metadata for first subgrid
    const Metadata &m0 = metadata[0];

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset_global = m.time_index - m0.time_index;
    const unsigned int station2     = m.baseline.station2;
    const unsigned int nr_timesteps = m.nr_timesteps;

    // Iterate timesteps
    int current_nr_timesteps = 0;
    for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps) {
        int aterm_idx = aterm_indices[time_offset_global + time_offset_local];

        // Determine number of timesteps to process
        current_nr_timesteps = 0;
        for (int time = time_offset_local; time < nr_timesteps; time++) {
            if (aterm_indices[time_offset_global + time] == aterm_idx) {
                current_nr_timesteps++;
            } else {
                break;
            }
        }

        // Iterate batch of visibilities from the same timeslot
        for (int i = tid; i < ALIGN(current_nr_timesteps * nr_channels, nr_threads); i += nr_threads) {
            int time = (i / nr_channels) + time_offset_local;
            int chan = (i % nr_channels);

            // Load UVW
            float u, v, w;
            if (time < nr_timesteps) {
                u = uvw[time_offset_global + time].u;
                v = uvw[time_offset_global + time].v;
                w = uvw[time_offset_global + time].w;
            }

            // Load wavenumber
            float wavenumber = wavenumbers[chan];

            // Accumulate sums in registers
            float2 sum_xx[current_nr_terms];
            float2 sum_xy[current_nr_terms];
            float2 sum_yx[current_nr_terms];
            float2 sum_yy[current_nr_terms];
            for (unsigned int term_nr = 0; term_nr < current_nr_terms; term_nr++) {
                sum_xx[term_nr] = make_float2(0, 0);
                sum_xy[term_nr] = make_float2(0, 0);
                sum_yx[term_nr] = make_float2(0, 0);
                sum_yy[term_nr] = make_float2(0, 0);
            }

            // Iterate all rows of the subgrid
            for (unsigned int y = 0; y < subgrid_size; y++) {
                __syncthreads();

                // Precompute data for one row
                for (unsigned j = tid; j < (subgrid_size*nr_terms); j += nr_threads) {
                    unsigned int term_nr = j / subgrid_size;
                    unsigned int x       = j % subgrid_size;

                    if (term_nr < nr_terms) {
                        // Compute shifted position in subgrid
                        unsigned int x_src = (x + (subgrid_size/2)) % subgrid_size;
                        unsigned int y_src = (y + (subgrid_size/2)) % subgrid_size;

                        // Load pixels
                        float2 pixel[NR_POLARIZATIONS];
                        for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            unsigned int pixel_idx = index_subgrid(subgrid_size, s, pol, y_src, x_src);
                            pixel[pol] = subgrid[pixel_idx];
                        }

                        // Load first aterm
                        size_t station1_idx = index_aterm(subgrid_size, nr_terms, aterm_idx, term_offset+term_nr, y, x);
                        float2 *aterm1 = (float2 *) &aterm_derivatives[station1_idx];

                        // Load second aterm
                        size_t station2_idx = index_aterm(subgrid_size, nr_stations, aterm_idx, station2, y, x);
                        float2 *aterm2 = (float2 *) &aterm[station2_idx];

                        // Apply aterm
                        apply_aterm_calibrate(pixel, aterm1, aterm2);

                        // Store pixels in shared memory
                        for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            pixels_[term_nr][pol][x] = pixel[pol];
                        }
                    } // end if
                } // end for j (subgrid_size * terms)

                __syncthreads();

                // Iterate all columns of current row
                for (unsigned int x = 0; x < subgrid_size; x++) {

                    // Load l,m,n
                    float l = lmnp_[y][x].x;
                    float m = lmnp_[y][x].y;
                    float n = lmnp_[y][x].z;

                    // Load phase offset
                    float phase_offset = lmnp_[y][x].w;

                    // Compute phase index
                    float phase_index = u*l + v*m + w*n;

                    // Compute phasor
                    float  phase  = (phase_index * wavenumber) - phase_offset;
                    float2 phasor = make_float2(raw_cos(phase), raw_sin(phase));

                    // Iterate all terms
                    for (unsigned int term_nr = 0; term_nr < current_nr_terms; term_nr++) {

                        // Load pixels
                        float2 pixel_xx = pixels_[term_nr][0][x];
                        float2 pixel_xy = pixels_[term_nr][1][x];
                        float2 pixel_yx = pixels_[term_nr][2][x];
                        float2 pixel_yy = pixels_[term_nr][3][x];

                        // Update sums
                        sum_xx[term_nr].x += phasor.x * pixel_xx.x;
                        sum_xx[term_nr].y += phasor.x * pixel_xx.y;
                        sum_xx[term_nr].x -= phasor.y * pixel_xx.y;
                        sum_xx[term_nr].y += phasor.y * pixel_xx.x;

                        sum_xy[term_nr].x += phasor.x * pixel_xy.x;
                        sum_xy[term_nr].y += phasor.x * pixel_xy.y;
                        sum_xy[term_nr].x -= phasor.y * pixel_xy.y;
                        sum_xy[term_nr].y += phasor.y * pixel_xy.x;

                        sum_yx[term_nr].x += phasor.x * pixel_yx.x;
                        sum_yx[term_nr].y += phasor.x * pixel_yx.y;
                        sum_yx[term_nr].x -= phasor.y * pixel_yx.y;
                        sum_yx[term_nr].y += phasor.y * pixel_yx.x;

                        sum_yy[term_nr].x += phasor.x * pixel_yy.x;
                        sum_yy[term_nr].y += phasor.x * pixel_yy.y;
                        sum_yy[term_nr].x -= phasor.y * pixel_yy.y;
                        sum_yy[term_nr].y += phasor.y * pixel_yy.x;
                    } // end for term_nr
                } // end for x
            } // end for y

            // Scale sums and store in device memory
            for (unsigned int term_nr = 0; term_nr < current_nr_terms; term_nr++) {
                const float scale = 1.0f / nr_pixels;
                if (time < nr_timesteps) {
                    unsigned int time_idx = time_offset_global + time;
                    unsigned int chan_idx = chan;
                    unsigned int sum_idx_xx = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, 0, time_idx, chan_idx);
                    unsigned int sum_idx_xy = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, 1, time_idx, chan_idx);
                    unsigned int sum_idx_yx = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, 2, time_idx, chan_idx);
                    unsigned int sum_idx_yy = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, 3, time_idx, chan_idx);
                    sum_deriv[sum_idx_xx] = sum_xx[term_nr] * scale;
                    sum_deriv[sum_idx_xy] = sum_xy[term_nr] * scale;
                    sum_deriv[sum_idx_yx] = sum_yx[term_nr] * scale;
                    sum_deriv[sum_idx_yy] = sum_yy[term_nr] * scale;
                }
            } // end for term_nr

            __syncthreads();

        } // end for i (visibilities)
    } // end for time_offset_local
} // end update_sums


__device__ void update_gradient(
    const int                         subgrid_size,
    const float                       image_size,
    const unsigned int                total_nr_timesteps,
    const unsigned int                nr_channels,
    const unsigned int                nr_stations,
    const unsigned int                nr_terms,
    const UVW*           __restrict__ uvw,
    const float2*        __restrict__ aterm,
    const int*           __restrict__ aterm_indices,
    const float2*        __restrict__ aterm_derivatives,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sum_aterm,
          float2*        __restrict__ sum_deriv,
          float2*        __restrict__ gradient,
          float4                      lmnp_[MAX_SUBGRID_SIZE][MAX_SUBGRID_SIZE])
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned nr_threads = blockDim.x * blockDim.y;
    unsigned s          = blockIdx.x;
    unsigned nr_pixels  = subgrid_size * subgrid_size;

    // Load metadata for first subgrid
    const Metadata &m0 = metadata[0];

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset_global = m.time_index - m0.time_index;
    const unsigned int nr_timesteps = m.nr_timesteps;
    const unsigned int station1     = m.baseline.station1;
    const unsigned int station2     = m.baseline.station2;

    // Shared memory
    __shared__ float2 pixels_[NR_POLARIZATIONS][MAX_SUBGRID_SIZE];

    // Iterate timesteps
    int current_nr_timesteps = 0;
    for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps) {
        int aterm_idx = aterm_indices[time_offset_global + time_offset_local];

        // Determine number of timesteps to process
        current_nr_timesteps = 0;
        for (int time = time_offset_local; time < nr_timesteps; time++) {
            if (aterm_indices[time_offset_global + time] == aterm_idx) {
                current_nr_timesteps++;
            } else {
                break;
            }
        }

        // Iterate batch of visibilities from the same timeslot
        for (int i = tid; i < ALIGN(current_nr_timesteps * nr_channels, nr_threads); i += nr_threads) {
            int time = (i / nr_channels) + time_offset_local;
            int chan = (i % nr_channels);

            // Load UVW
            float u, v, w;
            if (time < nr_timesteps) {
                u = uvw[time_offset_global + time].u;
                v = uvw[time_offset_global + time].v;
                w = uvw[time_offset_global + time].w;
            }

            // Load wavenumber
            float wavenumber = wavenumbers[chan];

            // Accumulate sums in registers
            float2 sum_xx = make_float2(0, 0);
            float2 sum_xy = make_float2(0, 0);
            float2 sum_yx = make_float2(0, 0);
            float2 sum_yy = make_float2(0, 0);

            // Iterate all rows of the subgrid
            for (unsigned int y = 0; y < subgrid_size; y++) {
                __syncthreads();

                // Precompute data for one row
                for (unsigned x = tid; x < subgrid_size; x += nr_threads) {

                    if (x < subgrid_size) {
                        // Compute shifted position in subgrid
                        unsigned int x_src = (x + (subgrid_size/2)) % subgrid_size;
                        unsigned int y_src = (y + (subgrid_size/2)) % subgrid_size;

                        // Load pixels
                        float2 pixel[NR_POLARIZATIONS];
                        for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            unsigned int pixel_idx = index_subgrid(subgrid_size, s, pol, y_src, x_src);
                            pixel[pol] = subgrid[pixel_idx];
                        }

                        // Load first aterm
                        size_t station1_idx = index_aterm(subgrid_size, nr_stations, aterm_idx, station1, y, x);
                        float2 *aterm1 = (float2 *) &aterm[station1_idx];

                        // Load second aterm
                        size_t station2_idx = index_aterm(subgrid_size, nr_stations, aterm_idx, station2, y, x);
                        float2 *aterm2 = (float2 *) &aterm[station2_idx];

                        // Apply aterm
                        apply_aterm_calibrate(pixel, aterm1, aterm2);

                        // Store pixels in shared memory
                        for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            pixels_[pol][x] = pixel[pol];
                        }
                    }
                } // end for x

                __syncthreads();

                // Iterate all columns of current row
                for (unsigned int x = 0; x < subgrid_size; x++) {

                    // Load l,m,n
                    float l = lmnp_[y][x].x;
                    float m = lmnp_[y][x].y;
                    float n = lmnp_[y][x].z;

                    // Load phase offset
                    float phase_offset = lmnp_[y][x].w;

                    // Compute phase index
                    float phase_index = u*l + v*m + w*n;

                    // Compute phasor
                    float  phase  = (phase_index * wavenumber) - phase_offset;
                    float2 phasor = make_float2(raw_cos(phase), raw_sin(phase));

                    // Load pixels
                    float2 pixel_xx = pixels_[0][x];
                    float2 pixel_xy = pixels_[1][x];
                    float2 pixel_yx = pixels_[2][x];
                    float2 pixel_yy = pixels_[3][x];

                    // Update sums
                    sum_xx.x += phasor.x * pixel_xx.x;
                    sum_xx.y += phasor.x * pixel_xx.y;
                    sum_xx.x -= phasor.y * pixel_xx.y;
                    sum_xx.y += phasor.y * pixel_xx.x;

                    sum_xy.x += phasor.x * pixel_xy.x;
                    sum_xy.y += phasor.x * pixel_xy.y;
                    sum_xy.x -= phasor.y * pixel_xy.y;
                    sum_xy.y += phasor.y * pixel_xy.x;

                    sum_yx.x += phasor.x * pixel_yx.x;
                    sum_yx.y += phasor.x * pixel_yx.y;
                    sum_yx.x -= phasor.y * pixel_yx.y;
                    sum_yx.y += phasor.y * pixel_yx.x;

                    sum_yy.x += phasor.x * pixel_yy.x;
                    sum_yy.y += phasor.x * pixel_yy.y;
                    sum_yy.x -= phasor.y * pixel_yy.y;
                    sum_yy.y += phasor.y * pixel_yy.x;
                } // end for x
            } // end for y

            // Compute sum_aterm
            const float scale = 1.0f / nr_pixels;

            if ((time - time_offset_local) < current_nr_timesteps) {

                // Iterate all terms
                for (unsigned term_nr = 0; term_nr < nr_terms; term_nr++) {

                    // Compute residual
                    unsigned int time_idx = time_offset_global + time;
                    unsigned int chan_idx = chan;
                    unsigned int vis_idx_xx = index_visibility(nr_channels, time_idx, chan_idx, 0);
                    unsigned int vis_idx_xy = index_visibility(nr_channels, time_idx, chan_idx, 1);
                    unsigned int vis_idx_yx = index_visibility(nr_channels, time_idx, chan_idx, 2);
                    unsigned int vis_idx_yy = index_visibility(nr_channels, time_idx, chan_idx, 3);
                    float2 residual_xx = (visibilities[vis_idx_xx] - (sum_xx * scale)) * weights[vis_idx_xx];
                    float2 residual_xy = (visibilities[vis_idx_xy] - (sum_xy * scale)) * weights[vis_idx_xy];
                    float2 residual_yx = (visibilities[vis_idx_yx] - (sum_yx * scale)) * weights[vis_idx_yx];
                    float2 residual_yy = (visibilities[vis_idx_yy] - (sum_yy * scale)) * weights[vis_idx_yy];

                    // Load derivative sums
                    unsigned int sum_deriv_idx_xx = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, 0, time_idx, chan_idx);
                    unsigned int sum_deriv_idx_xy = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, 1, time_idx, chan_idx);
                    unsigned int sum_deriv_idx_yx = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, 2, time_idx, chan_idx);
                    unsigned int sum_deriv_idx_yy = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, 3, time_idx, chan_idx);
                    float2 sum_xx = sum_deriv[sum_deriv_idx_xx];
                    float2 sum_xy = sum_deriv[sum_deriv_idx_xy];
                    float2 sum_yx = sum_deriv[sum_deriv_idx_yx];
                    float2 sum_yy = sum_deriv[sum_deriv_idx_yy];

                    // Compute gradient update
                    float2 update = make_float2(0, 0);

                    update.x += sum_xx.x * residual_xx.x;
                    update.x += sum_xx.y * residual_xx.y;
                    update.y += sum_xx.x * residual_xx.y;
                    update.y -= sum_xx.y * residual_xx.x;

                    update.x += sum_xy.x * residual_xy.x;
                    update.x += sum_xy.y * residual_xy.y;
                    update.y += sum_xy.x * residual_xy.y;
                    update.y -= sum_xy.y * residual_xy.x;

                    update.x += sum_yx.x * residual_yx.x;
                    update.x += sum_yx.y * residual_yx.y;
                    update.y += sum_yx.x * residual_yx.y;
                    update.y -= sum_yx.y * residual_yx.x;

                    update.x += sum_yy.x * residual_yy.x;
                    update.x += sum_yy.y * residual_yy.y;
                    update.y += sum_yy.x * residual_yy.y;
                    update.y -= sum_yy.y * residual_yy.x;

                    // Update gradient
                    unsigned int idx = aterm_idx * nr_terms + term_nr;
                    atomicAdd(&gradient[idx], update);

                } // end for term
            } // end if time

            __syncthreads();

        } // end for i (visibilities)
    }  // end for time_offset_local
} // end update_gradient


__device__ void update_hessian(
    const unsigned int                total_nr_timesteps,
    const unsigned int                nr_channels,
    const unsigned int                nr_stations,
    const unsigned int                nr_terms,
    const int*           __restrict__ aterm_indices,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const Metadata*      __restrict__ metadata,
          float2*        __restrict__ sum_aterm,
          float2*        __restrict__ sum_deriv,
          float2*        __restrict__ hessian)
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned s          = blockIdx.x;
    unsigned nr_threads = blockDim.x * blockDim.y;

    // Metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset_global = m.time_index;
    const unsigned int nr_timesteps = m.nr_timesteps;

    // Iterate timesteps
    int current_nr_timesteps = 0;
    for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps) {
        int aterm_idx = aterm_indices[time_offset_global + time_offset_local];

        // Determine number of timesteps to process
        current_nr_timesteps = 0;
        for (int time = time_offset_local; time < nr_timesteps; time++) {
            if (aterm_indices[time_offset_global + time] == aterm_idx) {
                current_nr_timesteps++;
            } else {
                break;
            }
        }

        // Iterate all terms * terms
        for (unsigned int term_nr = tid; term_nr < (nr_terms*nr_terms); term_nr += nr_threads) {
            unsigned term_nr1 = term_nr / nr_terms;
            unsigned term_nr0 = term_nr % nr_terms;

            // Compute hessian update
            float2 update = make_float2(0, 0);

            // Iterate all timesteps
            for (unsigned int time = 0; time < current_nr_timesteps; time++) {

                // Iterate all channels
                for (unsigned int chan = 0; chan < nr_channels; chan++) {

                    // Iterate all polarizations
                    for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                        unsigned int time_idx = time_offset_global + time_offset_local + time;
                        unsigned int chan_idx = chan;
                        unsigned int  vis_idx = index_visibility(nr_channels, time_idx, chan_idx, pol);
                        unsigned int sum_idx0 = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr0, pol, time_idx, chan_idx);
                        unsigned int sum_idx1 = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr1, pol, time_idx, chan_idx);
                        float2 sum0 = sum_deriv[sum_idx0];
                        float2 sum1 = sum_deriv[sum_idx1] * weights[vis_idx];

                        // Update hessian
                        if (term_nr0 < nr_terms) {
                            update.x += sum1.x * sum0.x;
                            update.x += sum1.y * sum0.y;
                            update.y += sum1.y * sum0.x;
                            update.y -= sum1.x * sum0.y;
                        }
                    } // end for pol
                } // end chan
            } // end for time

            __syncthreads();

            // Update hessian
            if (term_nr0 < nr_terms) {
                unsigned idx = aterm_idx * nr_terms * nr_terms + term_nr1 * nr_terms + term_nr0;
                atomicAdd(&hessian[idx], update);
            }
        } // end for term_nr (terms * terms)
    } // end for time_offset_local
} // end update_hessian


#define UPDATE_SUMS(current_nr_terms) \
    for (; (term_offset + current_nr_terms) <= nr_terms; term_offset += current_nr_terms) { \
        update_sums<current_nr_terms>( \
                subgrid_size, image_size, total_nr_timesteps, nr_channels, nr_stations, \
                nr_terms, term_offset, \
                uvw, aterm, aterm_indices, aterm_derivatives, wavenumbers, \
                visibilities, weights, metadata, subgrid, \
                sum_aterm, sum_deriv, lmnp_, pixels_); \
    }

extern "C" {

__global__ void kernel_calibrate_sums(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         total_nr_timesteps,
    const int                         nr_channels,
    const int                         nr_stations,
    const int                         nr_terms,
    const UVW*           __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const int*           __restrict__ aterm_indices,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sum_aterm,
          float2*        __restrict__ sum_deriv,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    // Shared memory
    __shared__ float4 lmnp_[MAX_SUBGRID_SIZE][MAX_SUBGRID_SIZE];
    __shared__ float2 pixels_[MAX_NR_TERMS][NR_POLARIZATIONS][MAX_SUBGRID_SIZE];

    compute_lmnp(grid_size, subgrid_size, image_size, w_step, metadata, lmnp_);

    int term_offset = 0;
    UPDATE_SUMS(8)
    UPDATE_SUMS(7)
    UPDATE_SUMS(6)
    UPDATE_SUMS(5)
    UPDATE_SUMS(4)
    UPDATE_SUMS(3)
    UPDATE_SUMS(2)
    UPDATE_SUMS(1)
} // end kernel_calibrate_sums


__global__ void kernel_calibrate_gradient(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         total_nr_timesteps,
    const int                         nr_channels,
    const int                         nr_stations,
    const int                         nr_terms,
    const UVW*           __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const int*           __restrict__ aterm_indices,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sum_aterm,
          float2*        __restrict__ sum_deriv,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    // Shared memory
    __shared__ float4 lmnp_[MAX_SUBGRID_SIZE][MAX_SUBGRID_SIZE];

    compute_lmnp(grid_size, subgrid_size, image_size, w_step, metadata, lmnp_);

    update_gradient(
        subgrid_size, image_size, total_nr_timesteps,
        nr_channels, nr_stations, nr_terms,
        uvw, aterm, aterm_indices, aterm_derivatives,
        wavenumbers, visibilities, weights, metadata, subgrid,
        sum_aterm, sum_deriv, gradient, lmnp_);
} // end kernel_calibrate_gradient


__global__ void kernel_calibrate_hessian(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         total_nr_timesteps,
    const int                         nr_channels,
    const int                         nr_stations,
    const int                         nr_terms,
    const UVW*           __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const int*           __restrict__ aterm_indices,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sum_aterm,
          float2*        __restrict__ sum_deriv,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    update_hessian(
        total_nr_timesteps, nr_channels, nr_stations, nr_terms,
        aterm_indices, visibilities, weights, metadata, sum_aterm, sum_deriv, hessian);
} // end kernel_calibrate_hessian

} // end extern "C"
