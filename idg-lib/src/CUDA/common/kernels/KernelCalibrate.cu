#include "hip/hip_runtime.h"
#include "Types.h"
#include "math.cu"

extern "C" {

/*
    Kernel
*/
__global__ void kernel_calibrate(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         nr_channels,
    const int                         nr_terms,
    const UVW*           __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    int s          = blockIdx.x;
    int tidx       = threadIdx.x;
    int tidy       = threadIdx.y;
    int tid        = tidx + tidy * blockDim.x;
    int nr_threads = blockDim.x * blockDim.y;

    // Load metadata for first subgrid
    const Metadata &m_0 = metadata[0];

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const int time_offset_global = (m.baseline_offset - m_0.baseline_offset) + m.time_offset;
    const int nr_timesteps = m.nr_timesteps;
    const int x_coordinate = m.coordinate.x;
    const int y_coordinate = m.coordinate.y;
} // end kernel_calibrate

} // end extern "C"
