#include "hip/hip_runtime.h"
#include "Types.h"
#include "math.cu"

#define ALIGN(N,A) (((N)+(A)-1)/(A)*(A))
#define MAX_NR_TERMS     8
#define BATCH_SIZE_PIXELS     256

inline __device__ long index_sums(
    unsigned int total_nr_timesteps, // number of timesteps for all baselines
    unsigned int nr_channels,        // number channels for a single baseline
    unsigned int term_nr,
    unsigned int pol,
    unsigned int time,
    unsigned int chan)
{
    // sums: [MAX_NR_TERMS][NR_POLARIZATIONS][TOTAL_NR_TIMESTEPS][NR_CHANNELS]
    return term_nr * NR_POLARIZATIONS * total_nr_timesteps * nr_channels +
           pol * total_nr_timesteps * nr_channels +
           time * nr_channels +
           chan;
}

inline __device__ float4 compute_lmnp(
    const Coordinate coordinate,
    const int        y,
    const int        x,
    const int        grid_size,
    const int        subgrid_size,
    const float      image_size,
    const float      w_step)
{
    // Locateion of current subgrid
    const int x_coordinate = coordinate.x;
    const int y_coordinate = coordinate.y;
    const int z_coordinate = coordinate.z;

    // Compute u,v,w offset in wavelenghts
    const float u_offset = (x_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
    const float v_offset = (y_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
    const float w_offset = w_step * ((float) z_coordinate + 0.5) * 2 * M_PI;

    // Compute l,m,n and phase_offset
    float l = compute_l(x, subgrid_size, image_size);
    float m = compute_m(y, subgrid_size, image_size);
    float n = compute_n(l, m);
    float phase_offset = u_offset*l + v_offset*m + w_offset*n;

    // Return result
    return make_float4(l, m, n, phase_offset);
} // end compute_lmnp


__device__ void update_sums(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const unsigned int                total_nr_timesteps,
    const unsigned int                nr_channels,
    const unsigned int                nr_stations,
    const unsigned int                nr_terms,
    const UVW<float>*    __restrict__ uvw,
    const float2*        __restrict__ aterm,
    const int*           __restrict__ aterm_indices,
    const float2*        __restrict__ aterm_derivatives,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ gradient)
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned nr_threads = blockDim.x * blockDim.y;
    unsigned s          = blockIdx.x;
    unsigned nr_pixels  = subgrid_size * subgrid_size;

    // Load metadata for first subgrid
    const Metadata &m0 = metadata[0];

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset_global = m.time_index - m0.time_index;
    const unsigned int station1     = m.baseline.station1;
    const unsigned int station2     = m.baseline.station2;
    const unsigned int nr_timesteps = m.nr_timesteps;
    const Coordinate coordinate     = m.coordinate;

    // Shared memory
    __shared__ float4 lmnp_[BATCH_SIZE_PIXELS];
    __shared__ float2 pixels_[NR_POLARIZATIONS][BATCH_SIZE_PIXELS];

    // Iterate all terms
    for (unsigned int term_nr = 0; term_nr < nr_terms; term_nr++) {

        // Iterate timesteps
        int current_nr_timesteps = 0;
        for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps) {
            int aterm_idx = aterm_indices[time_offset_global + time_offset_local];

            // Determine number of timesteps to process
            current_nr_timesteps = 0;
            for (int time = time_offset_local; time < nr_timesteps; time++) {
                if (aterm_indices[time_offset_global + time] == aterm_idx) {
                    current_nr_timesteps++;
                } else {
                    break;
                }
            }

            // Iterate batch of visibilities from the same timeslot
            for (int i = tid; i < ALIGN(current_nr_timesteps * nr_channels, nr_threads); i += nr_threads) {
                unsigned int time_idx_batch  = (i / nr_channels);
                unsigned int chan_idx_local  = (i % nr_channels);
                unsigned int time_idx_local  = time_offset_local + time_idx_batch;
                unsigned int time_idx_global = time_offset_global + time_idx_local;

                float2 visibility[NR_POLARIZATIONS];

                // Load UVW
                float u, v, w;
                if (time_idx_batch < current_nr_timesteps) {
                    u = uvw[time_idx_global].u;
                    v = uvw[time_idx_global].v;
                    w = uvw[time_idx_global].w;
                }

                // Load wavenumber
                float wavenumber = wavenumbers[chan_idx_local];

                // Accumulate sums in registers
                float2 sum1[NR_POLARIZATIONS] = {0, 0};
                float2 sum2[NR_POLARIZATIONS] = {0, 0};

                // Iterate all pixels
                for (unsigned int pixel_offset = 0; pixel_offset < nr_pixels; pixel_offset += BATCH_SIZE_PIXELS) {
                    __syncthreads();

                    for (unsigned int j = tid; j < BATCH_SIZE_PIXELS; j += nr_threads) {
                        unsigned int y = (pixel_offset + j) / subgrid_size;
                        unsigned int x = (pixel_offset + j) % subgrid_size;

                        // Reset pixel to zero
                        for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            pixels_[pol][j] = make_float2(0, 0);
                        }

                        // Prepare batch
                        if (y < subgrid_size) {
                            // Compute shifted position in subgrid
                            unsigned int x_src = (x + (subgrid_size/2)) % subgrid_size;
                            unsigned int y_src = (y + (subgrid_size/2)) % subgrid_size;

                            // Load pixel and aterms
                            float2 pixel[NR_POLARIZATIONS];
                            float2 aterm1[NR_POLARIZATIONS];
                            float2 aterm2[NR_POLARIZATIONS];
                            for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                                unsigned int term_idx   = term_nr;
                                unsigned int pixel_idx  = index_subgrid(subgrid_size, s, pol, y_src, x_src);
                                unsigned int aterm1_idx = index_aterm_transposed(subgrid_size, nr_terms, aterm_idx, term_idx, y, x, pol);
                                unsigned int aterm2_idx = index_aterm_transposed(subgrid_size, nr_stations, aterm_idx, station2, y, x, pol);
                                pixel[pol]  = subgrid[pixel_idx];
                                aterm1[pol] = aterm_derivatives[aterm1_idx];
                                aterm2[pol] = aterm[aterm2_idx];
                            }

                            // Apply aterm
                            apply_aterm_calibrate(pixel, aterm1, aterm2);

                            // Store pixel in shared memory
                            for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                                pixels_[pol][j] = pixel[pol];
                            }

                            lmnp_[j] = compute_lmnp(coordinate, y, x, grid_size, subgrid_size, image_size, w_step);
                        }
                    } // end for j

                    __syncthreads();

                    // Iterate batch
                    for (unsigned int j = 0; j < BATCH_SIZE_PIXELS; j++) {
                        // Load l,m,n
                        float l = lmnp_[j].x;
                        float m = lmnp_[j].y;
                        float n = lmnp_[j].z;

                        // Load phase offset
                        float phase_offset = lmnp_[j].w;

                        // Compute phase index
                        float phase_index = u*l + v*m + w*n;

                        // Compute phasor
                        float  phase  = (phase_index * wavenumber) - phase_offset;
                        float2 phasor = make_float2(raw_cos(phase), raw_sin(phase));

                        // Update sums
                        for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            sum1[pol] += (phasor * pixels_[pol][j]);
                        }
                    } // end for j (batch)
                } // end for pixel_offset

                const float scale = 1.0f / nr_pixels;
                if (time_idx_batch < current_nr_timesteps) {
                    for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                        visibility[pol] = conj(sum1[pol]) * scale;
                    }
                }

                __syncthreads();

                // Iterate all pixels
                for (unsigned int pixel_offset = 0; pixel_offset < nr_pixels; pixel_offset += BATCH_SIZE_PIXELS) {
                    __syncthreads();

                    for (unsigned int j = tid; j < BATCH_SIZE_PIXELS; j += nr_threads) {
                        unsigned int y = (pixel_offset + j) / subgrid_size;
                        unsigned int x = (pixel_offset + j) % subgrid_size;

                        // Reset pixel to zero
                        for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            pixels_[pol][j] = make_float2(0, 0);
                        }

                        // Prepare batch
                        if (y < subgrid_size) {
                            // Compute shifted position in subgrid
                            unsigned int x_src = (x + (subgrid_size/2)) % subgrid_size;
                            unsigned int y_src = (y + (subgrid_size/2)) % subgrid_size;

                            // Load pixel and aterms
                            float2 pixel[NR_POLARIZATIONS];
                            float2 aterm1[NR_POLARIZATIONS];
                            float2 aterm2[NR_POLARIZATIONS];
                            for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                                unsigned int pixel_idx  = index_subgrid(subgrid_size, s, pol, y_src, x_src);
                                unsigned int aterm1_idx = index_aterm_transposed(subgrid_size, nr_stations, aterm_idx, station1, y, x, pol);
                                unsigned int aterm2_idx = index_aterm_transposed(subgrid_size, nr_stations, aterm_idx, station2, y, x, pol);
                                pixel[pol]  = subgrid[pixel_idx];
                                aterm1[pol] = aterm[aterm1_idx];
                                aterm2[pol] = aterm[aterm2_idx];
                            }

                            // Apply aterm
                            apply_aterm_calibrate(pixel, aterm1, aterm2);

                            // Store pixel in shared memory
                            for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                                pixels_[pol][j] = pixel[pol];
                            }

                            lmnp_[j] = compute_lmnp(coordinate, y, x, grid_size, subgrid_size, image_size, w_step);
                        }
                    } // end for j

                    __syncthreads();

                    // Iterate batch
                    for (unsigned int j = 0; j < BATCH_SIZE_PIXELS; j++) {
                        // Load l,m,n
                        float l = lmnp_[j].x;
                        float m = lmnp_[j].y;
                        float n = lmnp_[j].z;

                        // Load phase offset
                        float phase_offset = lmnp_[j].w;

                        // Compute phase index
                        float phase_index = u*l + v*m + w*n;

                        // Compute phasor
                        float  phase  = (phase_index * wavenumber) - phase_offset;
                        float2 phasor = make_float2(raw_cos(phase), raw_sin(phase));

                        for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            sum2[pol] += phasor * pixels_[pol][j];
                        }
                    } // end for j (batch)
                } // end for pixel_offset

                if (time_idx_batch < current_nr_timesteps) {

                    // Compute residual
                    float2 residual[NR_POLARIZATIONS];
                    for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                        unsigned int vis_idx = index_visibility(nr_channels, time_idx_global, chan_idx_local, pol);
                        residual[pol] = (visibilities[vis_idx] - (sum2[pol] * scale)) * weights[vis_idx];
                    }

                    // Compute gradient update
                    float2 update = make_float2(0, 0);

                    for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                        update += residual[pol] * visibility[pol];
                    }

                    // Update gradient
                    unsigned int idx = aterm_idx * nr_terms + term_nr;
                    atomicAdd(&gradient[idx], update);
                } // end if time

                __syncthreads();

            } // end for i (visibilities)
        } // end for time_offset_local
    } // end for term_nr
} // end update_sums


__device__ void update_hessian(
    const unsigned int                total_nr_timesteps,
    const unsigned int                nr_channels,
    const unsigned int                nr_stations,
    const unsigned int                nr_terms,
    const int*           __restrict__ aterm_indices,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const Metadata*      __restrict__ metadata,
          float2*        __restrict__ sums,
          float2*        __restrict__ hessian)
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned s          = blockIdx.x;
    unsigned nr_threads = blockDim.x * blockDim.y;

    // Metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset_global = m.time_index;
    const unsigned int nr_timesteps = m.nr_timesteps;

    // Iterate timesteps
    int current_nr_timesteps = 0;
    for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps) {
        int aterm_idx = aterm_indices[time_offset_global + time_offset_local];

        // Determine number of timesteps to process
        current_nr_timesteps = 0;
        for (int time = time_offset_local; time < nr_timesteps; time++) {
            if (aterm_indices[time_offset_global + time] == aterm_idx) {
                current_nr_timesteps++;
            } else {
                break;
            }
        }

        // Iterate all terms * terms
        for (unsigned int term_nr = tid; term_nr < (nr_terms*nr_terms); term_nr += nr_threads) {
            unsigned term_nr1 = term_nr / nr_terms;
            unsigned term_nr0 = term_nr % nr_terms;

            // Compute hessian update
            float2 update = make_float2(0, 0);

            // Iterate all timesteps
            for (unsigned int time = 0; time < current_nr_timesteps; time++) {

                // Iterate all channels
                for (unsigned int chan = 0; chan < nr_channels; chan++) {

                    // Iterate all polarizations
                    for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                        unsigned int time_idx_global = time_offset_global + time_offset_local + time;
                        unsigned int chan_idx = chan;
                        unsigned int  vis_idx = index_visibility(nr_channels, time_idx_global, chan_idx, pol);
                        unsigned int sum_idx0 = index_sums(total_nr_timesteps, nr_channels, term_nr0, pol, time_idx_global, chan_idx);
                        unsigned int sum_idx1 = index_sums(total_nr_timesteps, nr_channels, term_nr1, pol, time_idx_global, chan_idx);
                        float2 sum0 = sums[sum_idx0];
                        float2 sum1 = conj(sums[sum_idx1]) * weights[vis_idx];

                        // Update hessian
                        if (term_nr0 < nr_terms) {
                            update += sum0 * sum1;
                        }
                    } // end for pol
                } // end chan
            } // end for time

            __syncthreads();

            // Update hessian
            if (term_nr0 < nr_terms) {
                unsigned idx = aterm_idx * nr_terms * nr_terms + term_nr1 * nr_terms + term_nr0;
                atomicAdd(&hessian[idx], update);
            }
        } // end for term_nr (terms * terms)
    } // end for time_offset_local
} // end update_hessian


extern "C" {

__global__ void kernel_calibrate_sums(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         total_nr_timesteps,
    const int                         nr_channels,
    const int                         nr_stations,
    const int                         nr_terms,
    const UVW<float>*    __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const int*           __restrict__ aterm_indices,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sums,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    update_sums(
        grid_size, subgrid_size, image_size, w_step,
        total_nr_timesteps, nr_channels, nr_stations, nr_terms,
        uvw, aterm, aterm_indices, aterm_derivatives, wavenumbers,
        visibilities, weights, metadata, subgrid, gradient);
} // end kernel_calibrate_sums


__global__ void kernel_calibrate_hessian(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         total_nr_timesteps,
    const int                         nr_channels,
    const int                         nr_stations,
    const int                         nr_terms,
    const UVW<float>*    __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const int*           __restrict__ aterm_indices,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sums,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    update_hessian(
        total_nr_timesteps, nr_channels, nr_stations, nr_terms,
        aterm_indices, visibilities, weights, metadata, sums, hessian);
} // end kernel_calibrate_hessian

} // end extern "C"
