#include "hip/hip_runtime.h"
#include "Types.h"
#include "math.cu"

#define ALIGN(N,A) (((N)+(A)-1)/(A)*(A))
#define MAX_NR_TERMS     8
#define MAX_SUBGRID_SIZE 32
#define MAX_NR_THREADS   128

// Index in scratch_sum
inline __device__ long index_sums(
    unsigned int nr_timesteps,
    unsigned int nr_channels,
    unsigned int s,
    unsigned int time,
    unsigned int chan,
    unsigned int pol,
    unsigned int term_nr)
{
    // sums: [nr_subgrids][nr_timesteps][nr_channels][NR_TERMS][NR_POLARIZATIONS]
    return s * nr_timesteps * nr_channels * MAX_NR_TERMS * NR_POLARIZATIONS +
           time * nr_channels * MAX_NR_TERMS * NR_POLARIZATIONS +
           chan * MAX_NR_TERMS * NR_POLARIZATIONS +
           term_nr * NR_POLARIZATIONS +
           pol;
}

extern "C" {

__global__ void kernel_calibrate(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         nr_channels,
    const int                         nr_terms,
    const UVW*           __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ scratch_sum,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    unsigned s          = blockIdx.x;
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned nr_threads = blockDim.x * blockDim.y;
    unsigned nr_pixels  = subgrid_size * subgrid_size;

    // Find offset of first subgrid
    const Metadata &m_0       = metadata[0];

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset = (m.baseline_offset - m_0.baseline_offset) + m.time_offset;
    const unsigned int station1 = m.baseline.station1;
    const unsigned int station2 = m.baseline.station2;
    const int nr_timesteps      = m.nr_timesteps;
    const int x_coordinate      = m.coordinate.x;
    const int y_coordinate      = m.coordinate.y;
    const int z_coordinate      = m.coordinate.z;

    // Compute u,v,w offset in wavelenghts
    const float u_offset = (x_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
    const float v_offset = (y_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
    const float w_offset = w_step * ((float) z_coordinate + 0.5) * 2 * M_PI;

    __shared__ float4 lmn_[MAX_SUBGRID_SIZE];
    __shared__ float2 pixels_[NR_POLARIZATIONS][MAX_SUBGRID_SIZE][MAX_NR_TERMS];
    __shared__ float2 sums_[MAX_NR_THREADS][NR_POLARIZATIONS][MAX_NR_TERMS];
    __shared__ float2 gradient_[MAX_NR_TERMS];
    __shared__ float2 hessian_[MAX_NR_TERMS][MAX_NR_TERMS];

    /*
        Phase 0: initialize shared memory to zero
    */

    for (unsigned int i = tid; i < (NR_POLARIZATIONS*MAX_SUBGRID_SIZE*MAX_NR_TERMS); i += nr_threads) {
        pixels_[0][0][i] = make_float2(0, 0);
    }

    for (unsigned int i = tid; i < (MAX_NR_TERMS*MAX_NR_TERMS); i += nr_threads) {
        if (i < MAX_NR_TERMS) {
            gradient_[i] = make_float2(0, 0);
        }

        if (i < (MAX_NR_TERMS*MAX_NR_TERMS)) {
            hessian_[0][i] = make_float2(0, 0);
        }
    }

    for (unsigned int i = tid; i < (MAX_NR_THREADS*NR_POLARIZATIONS*MAX_NR_TERMS); i += nr_threads) {
        sums_[0][0][i] = make_float2(0, 0);
    }

    __syncthreads();

    // Iterate all timesteps
    for (unsigned int i = tid; i < ALIGN(nr_timesteps*nr_channels, nr_threads); i += nr_threads) {
        unsigned int time = i / nr_channels;
        unsigned int chan = i % nr_channels;

        /*
            Phase 1: "degrid" all subgrids, row by row
        */

        // Load UVW
        float u, v, w;
        if (time < nr_timesteps) {
            u = uvw[time_offset + time].u;
            v = uvw[time_offset + time].v;
            w = uvw[time_offset + time].w;
        }

        // Load wavenumber
        float wavenumber = wavenumbers[chan];

        // Accumulate sums in registers
        float2 sumXX[MAX_NR_TERMS];
        float2 sumXY[MAX_NR_TERMS];
        float2 sumYX[MAX_NR_TERMS];
        float2 sumYY[MAX_NR_TERMS];
        for (unsigned int term_nr = 0; term_nr < (nr_terms+1); term_nr++) {
            sumXX[term_nr] = make_float2(0, 0);
            sumXY[term_nr] = make_float2(0, 0);
            sumYX[term_nr] = make_float2(0, 0);
            sumYY[term_nr] = make_float2(0, 0);
        }

        // Iterate all rows of the subgrid
        for (unsigned int y = 0; y < subgrid_size; y++) {
            __syncthreads();

            // Precompute data for one row
            for (unsigned x = tid; x < subgrid_size; x += nr_threads) {

                if (x < subgrid_size) {
                    // Precompute l,m,n and phase offset
                    float l = compute_l(x, subgrid_size, image_size);
                    float m = compute_m(y, subgrid_size, image_size);
                    float n = compute_n(l, m);
                    float phase_offset = u_offset*l + v_offset*m + w_offset*n;
                    lmn_[x] = make_float4(l, m, n, phase_offset);

                    // Precompute pixels
                    for (unsigned term_nr = 0; term_nr < (nr_terms+1); term_nr++) {
                        // Compute shifted position in subgrid
                        unsigned int x_src = (x + (subgrid_size/2)) % subgrid_size;
                        unsigned int y_src = (y + (subgrid_size/2)) % subgrid_size;

                        // Load pixels
                        unsigned int pixel_idx_xx = index_subgrid(subgrid_size, s, 0, y_src, x_src);
                        unsigned int pixel_idx_xy = index_subgrid(subgrid_size, s, 1, y_src, x_src);
                        unsigned int pixel_idx_yx = index_subgrid(subgrid_size, s, 2, y_src, x_src);
                        unsigned int pixel_idx_yy = index_subgrid(subgrid_size, s, 3, y_src, x_src);
                        float2 pixelXX = subgrid[pixel_idx_xx];
                        float2 pixelXY = subgrid[pixel_idx_xy];
                        float2 pixelYX = subgrid[pixel_idx_yx];
                        float2 pixelYY = subgrid[pixel_idx_yy];

                        // Load first aterm
                        float2 aXX1, aXY1, aYX1, aYY1;

                        if (term_nr == nr_terms) {
                            // Load aterm for station1
                            size_t station1_idx = index_aterm(subgrid_size, 0, 0, station1, y, x);
                            aXX1 = aterm[station1_idx + 0];
                            aXY1 = aterm[station1_idx + 1];
                            aYX1 = aterm[station1_idx + 2];
                            aYY1 = aterm[station1_idx + 3];
                        } else {
                            // Load aterm derivative
                            size_t station1_idx = index_aterm(subgrid_size, 0, 0, term_nr, y, x);
                            aXX1 = aterm_derivatives[station1_idx + 0];
                            aXY1 = aterm_derivatives[station1_idx + 1];
                            aYX1 = aterm_derivatives[station1_idx + 2];
                            aYY1 = aterm_derivatives[station1_idx + 3];
                        }

                        // Load second aterm
                        float2 aXX2, aXY2, aYX2, aYY2;
                        size_t station2_idx = index_aterm(subgrid_size, 0, 0, station2, y, x);
                        aXX2 = aterm[station2_idx + 0];
                        aXY2 = aterm[station2_idx + 1];
                        aYX2 = aterm[station2_idx + 2];
                        aYY2 = aterm[station2_idx + 3];

                        // Apply aterm
                        apply_aterm(
                            aXX1, aYX1, aXY1, aYY1,
                            aXX2, aYX2, aXY2, aYY2,
                            pixelXX, pixelXY, pixelYX, pixelYY);

                        // Store pixels in shared memory
                        pixels_[0][x][term_nr] = pixelXX;
                        pixels_[1][x][term_nr] = pixelXY;
                        pixels_[2][x][term_nr] = pixelYX;
                        pixels_[3][x][term_nr] = pixelYY;
                    } // end for terms
                } // end if
            } // end for x

            __syncthreads();

            // Iterate all columns of the subgrid
            for (unsigned int x = 0; x < subgrid_size; x++) {

                // Load l,m,n
                float l = lmn_[x].x;
                float m = lmn_[x].y;
                float n = lmn_[x].z;

                // Load phase offset
                float phase_offset = lmn_[x].w;

                // Compute phase index
                float phase_index = u*l + v*m + w*n;

                // Compute phasor
                float  phase  = (phase_index * wavenumber) - phase_offset;
                float2 phasor = make_float2(raw_cos(phase), raw_sin(phase));

                // Iterate all terms
                for (unsigned int term_nr = 0; term_nr < MAX_NR_TERMS; term_nr++) {

                    // Load pixels
                    float2 pixelXX = pixels_[0][x][term_nr];
                    float2 pixelXY = pixels_[1][x][term_nr];
                    float2 pixelYX = pixels_[2][x][term_nr];
                    float2 pixelYY = pixels_[3][x][term_nr];

                    // Update sums
                    sumXX[term_nr].x += phasor.x * pixelXX.x;
                    sumXX[term_nr].y += phasor.x * pixelXX.y;
                    sumXX[term_nr].x -= phasor.y * pixelXX.y;
                    sumXX[term_nr].y += phasor.y * pixelXX.x;

                    sumXY[term_nr].x += phasor.x * pixelXY.x;
                    sumXY[term_nr].y += phasor.x * pixelXY.y;
                    sumXY[term_nr].x -= phasor.y * pixelXY.y;
                    sumXY[term_nr].y += phasor.y * pixelXY.x;

                    sumYX[term_nr].x += phasor.x * pixelYX.x;
                    sumYX[term_nr].y += phasor.x * pixelYX.y;
                    sumYX[term_nr].x -= phasor.y * pixelYX.y;
                    sumYX[term_nr].y += phasor.y * pixelYX.x;

                    sumYY[term_nr].x += phasor.x * pixelYY.x;
                    sumYY[term_nr].y += phasor.x * pixelYY.y;
                    sumYY[term_nr].x -= phasor.y * pixelYY.y;
                    sumYY[term_nr].y += phasor.y * pixelYY.x;
                } // end for term_nr
            } // end for x
        } // end for y

        // Scale sums and store in device memory
        for (unsigned int term_nr = 0; term_nr < MAX_NR_TERMS; term_nr++) {
            const float scale = 1.0f / nr_pixels;
            sums_[tid][0][term_nr] = sumXX[term_nr] * scale;
            sums_[tid][1][term_nr] = sumXY[term_nr] * scale;
            sums_[tid][2][term_nr] = sumYX[term_nr] * scale;
            sums_[tid][3][term_nr] = sumYY[term_nr] * scale;
        } // end for term_nr

        __syncthreads();

        /*
            Phase 2: update local gradient and hessian
        */

        // Iterate all visibilities
        for (unsigned int v = 0; v < MAX_NR_THREADS; v++) {
            unsigned int k = (i - tid) + v;
            unsigned int time = k / nr_channels;
            unsigned int chan = k % nr_channels;

            // Compute residual visibility
            float2 visibility_res[NR_POLARIZATIONS];
            for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                unsigned int time_idx = time_offset + time;
                unsigned int chan_idx = chan;
                unsigned int vis_idx  = index_visibility(nr_channels, time_idx, chan_idx, pol);
                if (time < nr_timesteps) {
                    visibility_res[pol] = visibilities[vis_idx + pol] - sums_[v][pol][nr_terms];
                }
            }

            // Iterate all terms * terms
            for (unsigned int term_nr = tid; term_nr < (nr_terms*nr_terms); term_nr += nr_threads) {
                unsigned term_nr0 = term_nr / nr_terms;
                unsigned term_nr1 = term_nr % nr_terms;

                // Iterate all polarizations
                for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {

                    // Update local gradient
                    if (term_nr < nr_terms) {
                        gradient_[term_nr].x +=
                            sums_[v][pol][term_nr].x * visibility_res[pol].x +
                            sums_[v][pol][term_nr].y * visibility_res[pol].y;
                        gradient_[term_nr].y +=
                            sums_[v][pol][term_nr].x * visibility_res[pol].y -
                            sums_[v][pol][term_nr].y * visibility_res[pol].x;
                    }

                    // Update local hessian
                    if (term_nr < (nr_terms*nr_terms)) {
                        hessian_[term_nr1][term_nr0].x +=
                            sums_[v][pol][term_nr0].x * sums_[v][pol][term_nr1].x +
                            sums_[v][pol][term_nr0].y * sums_[v][pol][term_nr1].y;
                        hessian_[term_nr0][term_nr1].y +=
                            sums_[v][pol][term_nr0].x * sums_[v][pol][term_nr1].y -
                            sums_[v][pol][term_nr0].y * sums_[v][pol][term_nr1].x;
                    }
                } // end for pol
            } // end for i (terms * terms)
        } // end for v (visibilities)
    } // end for i (visibilities)

    __syncthreads();

    /*
        Phase 3: update global gradient and hessian
    */

    // Iterate all terms * terms
    for (unsigned int i = tid; i < (nr_terms*nr_terms); i += nr_threads) {
        unsigned term_nr0 = i / nr_terms;
        unsigned term_nr1 = i % nr_terms;

        if (i < nr_terms) {
            atomicAdd(&gradient[i], gradient_[i]);
        }

        if (i < (nr_terms*nr_terms)) {
            atomicAdd(&hessian[i], hessian_[term_nr1][term_nr0]);
        }
    } // end for i
} // end kernel_calibrate

} // end extern "C"
