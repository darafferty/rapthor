#include "hip/hip_runtime.h"
#include "Types.h"
#include "math.cu"

#define MAX_NR_TERMS 8

// Index in scratch_sum
inline __device__ long index_sums(
    unsigned int nr_timesteps,
    unsigned int nr_channels,
    unsigned int nr_terms,
    unsigned int s,
    unsigned int time,
    unsigned int chan,
    unsigned int pol,
    unsigned int term_nr)
{
    // sums: [nr_subgrids][nr_timesteps][nr_channels][NR_POLARIZATIONS][nr_terms]
    return s * nr_timesteps * nr_channels * NR_POLARIZATIONS * nr_terms +
           time * nr_channels * NR_POLARIZATIONS * nr_terms +
           chan * NR_POLARIZATIONS * nr_terms +
           pol * nr_terms +
           term_nr;
}

// Index in scratch_pix
inline __device__ long index_pixels(
    unsigned int nr_terms,
    unsigned int subgrid_size,
    unsigned int s,
    unsigned int term_nr,
    unsigned int pol,
    unsigned int y,
    unsigned int x)
{
    // pix: [nr_subgrids][nr_terms][NR_POLARIZATIONS][subgrid_size][subgrid_size]
    return s * nr_terms * NR_POLARIZATIONS * subgrid_size * subgrid_size +
           term_nr * NR_POLARIZATIONS * subgrid_size * subgrid_size +
           pol * subgrid_size * subgrid_size +
           y * subgrid_size +
           x;
}

extern "C" {

__global__ void kernel_calibrate(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         nr_channels,
    const int                         nr_terms,
    const UVW*           __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ scratch_pix,
          float2*        __restrict__ scratch_sum,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    unsigned s          = blockIdx.x;
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned nr_threads = blockDim.x * blockDim.y;
    unsigned nr_pixels  = subgrid_size * subgrid_size;

    // Find offset of first subgrid
    const Metadata &m_0       = metadata[0];

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset = (m.baseline_offset - m_0.baseline_offset) + m.time_offset;
    const unsigned int station1 = m.baseline.station1;
    const unsigned int station2 = m.baseline.station2;
    const int nr_timesteps      = m.nr_timesteps;
    const int x_coordinate      = m.coordinate.x;
    const int y_coordinate      = m.coordinate.y;
    const int z_coordinate      = m.coordinate.z;

    // Compute u,v,w offset in wavelenghts
    const float u_offset = (x_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
    const float v_offset = (y_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
    const float w_offset = w_step * ((float) z_coordinate + 0.5) * 2 * M_PI;

    /*
        Phase 1: apply aterm to subgrids and store prepared subgrid in device memory
    */

    // Apply aterm to subgrid
    for (unsigned i = tid; i < nr_pixels; i += nr_threads) {
        if (i < nr_pixels) {
            for (unsigned term_nr = 0; term_nr < (nr_terms+1); term_nr++) {
                unsigned int y = i / subgrid_size;
                unsigned int x = i % subgrid_size;

                // Compute shifted position in subgrid
                unsigned int x_src = (x + (subgrid_size/2)) % subgrid_size;
                unsigned int y_src = (y + (subgrid_size/2)) % subgrid_size;

                // Load pixels
                unsigned int pixel_idx_xx = index_subgrid(subgrid_size, s, 0, y_src, x_src);
                unsigned int pixel_idx_xy = index_subgrid(subgrid_size, s, 1, y_src, x_src);
                unsigned int pixel_idx_yx = index_subgrid(subgrid_size, s, 2, y_src, x_src);
                unsigned int pixel_idx_yy = index_subgrid(subgrid_size, s, 3, y_src, x_src);
                float2 pixelXX = subgrid[pixel_idx_xx];
                float2 pixelXY = subgrid[pixel_idx_xy];
                float2 pixelYX = subgrid[pixel_idx_yx];
                float2 pixelYY = subgrid[pixel_idx_yy];

                // Load first aterm
                float2 aXX1, aXY1, aYX1, aYY1;

                if (term_nr == nr_terms) {
                    // Load aterm for station1
                    size_t station1_idx = index_aterm(subgrid_size, 0, 0, station1, y, x);
                    aXX1 = aterm[station1_idx + 0];
                    aXY1 = aterm[station1_idx + 1];
                    aYX1 = aterm[station1_idx + 2];
                    aYY1 = aterm[station1_idx + 3];
                } else {
                    // Load aterm derivative
                    size_t station1_idx = index_aterm(subgrid_size, 0, 0, term_nr, y, x);
                    aXX1 = aterm_derivatives[station1_idx + 0];
                    aXY1 = aterm_derivatives[station1_idx + 1];
                    aYX1 = aterm_derivatives[station1_idx + 2];
                    aYY1 = aterm_derivatives[station1_idx + 3];
                }

                // Load second aterm
                float2 aXX2, aXY2, aYX2, aYY2;
                size_t station2_idx = index_aterm(subgrid_size, 0, 0, station2, y, x);
                aXX2 = aterm[station2_idx + 0];
                aXY2 = aterm[station2_idx + 1];
                aYX2 = aterm[station2_idx + 2];
                aYY2 = aterm[station2_idx + 3];

                // Apply aterm
                apply_aterm(
                    aXX1, aYX1, aXY1, aYY1,
                    aXX2, aYX2, aXY2, aYY2,
                    pixelXX, pixelXY, pixelYX, pixelYY);

                // Store pixels
                pixel_idx_xx = index_pixels(nr_terms+1, subgrid_size, s, term_nr, 0, y, x);
                pixel_idx_xy = index_pixels(nr_terms+1, subgrid_size, s, term_nr, 1, y, x);
                pixel_idx_yx = index_pixels(nr_terms+1, subgrid_size, s, term_nr, 2, y, x);
                pixel_idx_yy = index_pixels(nr_terms+1, subgrid_size, s, term_nr, 3, y, x);
                scratch_pix[pixel_idx_xx] = pixelXX;
                scratch_pix[pixel_idx_xy] = pixelXY;
                scratch_pix[pixel_idx_yx] = pixelYX;
                scratch_pix[pixel_idx_yy] = pixelYY;
            } // end for terms
        } // end if
    } // end for pixels

    __syncthreads();

    __shared__ float2 sums_[NR_POLARIZATIONS][MAX_NR_TERMS];
    __shared__ float2 gradient_[MAX_NR_TERMS];
    __shared__ float2 hessian_[MAX_NR_TERMS][MAX_NR_TERMS];

    // Initialize shared memory to zero
    for (unsigned int i = tid; i < nr_terms * nr_terms; i += nr_threads) {
        unsigned term_nr1 = i / nr_terms;
        unsigned term_nr0 = i % nr_terms;

        if (i < nr_terms) {
            gradient_[term_nr0] = make_float2(0, 0);

            for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                sums_[pol][term_nr0] = make_float2(0, 0);
            }
        }

        if (i < (nr_terms*nr_terms)) {
            hessian_[term_nr1][term_nr0] = make_float2(0, 0);
        }

    } // end for i

    __syncthreads();

    /*
        Phase 2: "degrid" all prepared subgrids, store results in local memory
    */

    // Iterate all timesteps
    for (unsigned int time = 0; time < nr_timesteps; time++) {

        // Load UVW
        float u = uvw[time_offset + time].u;
        float v = uvw[time_offset + time].v;
        float w = uvw[time_offset + time].w;

        // Iterate all channels
        for (unsigned int chan = 0; chan < nr_channels; chan++) {

            // Load wavenumber
            float wavenumber = wavenumbers[chan];

            // Iterate all terms
            for (unsigned int term_nr = 0; term_nr < (nr_terms+1); term_nr++) {

                // Accumulate sums in registers
                float2 sums[NR_POLARIZATIONS];
                for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                    sums[pol] = make_float2(0, 0);
                }

                // Iterate all pixels
                for (unsigned int j = 0; j < nr_pixels; j++) {
                    unsigned y = j / subgrid_size;
                    unsigned x = j % subgrid_size;

                    // Compute l,m,n
                    const float l = compute_l(x, subgrid_size, image_size);
                    const float m = compute_m(y, subgrid_size, image_size);
                    const float n = compute_n(l, m);

                    // Compute phase offset
                    float phase_offset = u_offset*l + v_offset*m + w_offset*n;

                    // Compute phase index
                    float phase_index = u*l + v*m + w*n;

                    // Compute phasor
                    float  phase  = (phase_index * wavenumber) - phase_offset;
                    float2 phasor = make_float2(raw_cos(phase), raw_sin(phase));

                    for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                        // Load pixel
                        unsigned int pixel_idx = index_pixels(nr_terms+1, subgrid_size, s, term_nr, pol, y, x);
                        float2 pixel = scratch_pix[pixel_idx];

                        // Update sum
                        sums[pol].x += phasor.x * pixel.x;
                        sums[pol].y += phasor.x * pixel.y;
                        sums[pol].x -= phasor.y * pixel.y;
                        sums[pol].y += phasor.y * pixel.x;
                    }
                } // end for j (pixels)

                // Scale sums
                const float scale = 1.0f / nr_pixels;
                for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                    unsigned int idx = index_sums(nr_timesteps, nr_channels, (nr_terms+1), s, time, chan, pol, term_nr);
                    scratch_sum[idx] = sums[pol] * scale;;
                }

            } // end for term_nr
        } // end for chan
    } // end for time

    __syncthreads();

    /*
        Phase 3: update local gradient and hessian
    */
    // Iterate all timesteps
    for (unsigned int time = 0; time < nr_timesteps; time++) {
        // Iterate all channels
        for (unsigned int chan = 0; chan < nr_channels; chan++) {

            // Load sums for current visibility
            for (unsigned int term_nr = 0; term_nr < (nr_terms+1); term_nr++) {
                for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                    unsigned int idx = index_sums(nr_timesteps, nr_channels, (nr_terms+1), s, time, chan, pol, term_nr);
                    sums_[pol][term_nr] = scratch_sum[idx];
                }
            }

            __syncthreads();

            // Compute residual visibility
            float2 visibility_res[NR_POLARIZATIONS];
            for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                unsigned int time_idx = time_offset + time;
                unsigned int chan_idx = chan;
                unsigned int vis_idx  = index_visibility(nr_channels, time_idx, chan_idx, pol);
                visibility_res[pol] = visibilities[vis_idx + pol] - sums_[pol][nr_terms];
            }

            // Iterate all terms * terms
            for (unsigned int i = tid; i < (nr_terms*nr_terms); i += nr_threads) {
                unsigned term_nr0 = i / nr_terms;
                unsigned term_nr1 = i % nr_terms;

                // Iterate all polarizations
                for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {

                    // Update local gradient
                    if (i < nr_terms) {
                        gradient_[i].x +=
                            sums_[pol][i].x * visibility_res[pol].x +
                            sums_[pol][i].y * visibility_res[pol].y;
                        gradient_[i].y +=
                            sums_[pol][i].x * visibility_res[pol].y -
                            sums_[pol][i].y * visibility_res[pol].x;
                    }

                    // Update local hessian
                    if (i < (nr_terms*nr_terms)) {
                        hessian_[term_nr1][term_nr0].x +=
                            sums_[pol][term_nr0].x * sums_[pol][term_nr1].x +
                            sums_[pol][term_nr0].y * sums_[pol][term_nr1].y;
                        hessian_[term_nr0][term_nr1].y +=
                            sums_[pol][term_nr0].x * sums_[pol][term_nr1].y -
                            sums_[pol][term_nr0].y * sums_[pol][term_nr1].x;
                    }
                } // end for pol
            } // end for i (terms * terms)
        } // end for chan
    } // end for time

    __syncthreads();

    /*
        Phase 4: update global gradient and hessian
    */

    // Iterate all terms * terms
    for (unsigned int i = tid; i < (nr_terms*nr_terms); i += nr_threads) {
        unsigned term_nr0 = i / nr_terms;
        unsigned term_nr1 = i % nr_terms;

        if (i < nr_terms) {
            atomicAdd(&gradient[i], gradient_[i]);
        }

        if (i < (nr_terms*nr_terms)) {
            atomicAdd(&hessian[i], hessian_[term_nr1][term_nr0]);
        }
    } // end for i
} // end kernel_calibrate

} // end extern "C"
