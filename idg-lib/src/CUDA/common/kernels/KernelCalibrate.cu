#include "hip/hip_runtime.h"
#include "Types.h"
#include "math.cu"

#define ALIGN(N,A) (((N)+(A)-1)/(A)*(A))
#define MAX_NR_TERMS     8
#define MAX_SUBGRID_SIZE 32
#define MAX_NR_THREADS   128
#define MAX_NR_TIMESTEPS 128

inline __device__ long index_sum_deriv(
    unsigned int total_nr_timesteps, // number of timesteps for all baselines
    unsigned int nr_channels,        // number channels for a single baseline
    unsigned int term_nr,
    unsigned int pol,
    unsigned int time,
    unsigned int chan)
{
    // sums: [MAX_NR_TERMS][NR_POLARIZATIONS][TOTAL_NR_TIMESTEPS][NR_CHANNELS]
    return term_nr * NR_POLARIZATIONS * total_nr_timesteps * nr_channels +
           pol * total_nr_timesteps * nr_channels +
           time * nr_channels +
           chan;
}

inline __device__ long index_sum_aterm(
    unsigned int total_nr_timesteps, // number of timesteps for all baselines
    unsigned int nr_channels,        // number channels for a single baseline
    unsigned int pol,
    unsigned int time,
    unsigned int chan)
{
    // sums: [NR_POLARIZATIONS][TOTAL_NR_TIMESTEPS][NR_CHANNELS]
    return pol * total_nr_timesteps * nr_channels +
           time * nr_channels +
           chan;
}

__device__ void compute_lmnp(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const Metadata*      __restrict__ metadata,
          float4                      lmnp_[MAX_SUBGRID_SIZE][MAX_SUBGRID_SIZE])
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned nr_threads = blockDim.x * blockDim.y;
    unsigned s          = blockIdx.x;

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const int x_coordinate      = m.coordinate.x;
    const int y_coordinate      = m.coordinate.y;
    const int z_coordinate      = m.coordinate.z;

    // Compute u,v,w offset in wavelenghts
    const float u_offset = (x_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
    const float v_offset = (y_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
    const float w_offset = w_step * ((float) z_coordinate + 0.5) * 2 * M_PI;

    for (unsigned int i = tid; i < (subgrid_size * subgrid_size); i += nr_threads) {
        unsigned int y = i / subgrid_size;
        unsigned int x = i % subgrid_size;

        if (y < subgrid_size) {
            float l = compute_l(x, subgrid_size, image_size);
            float m = compute_m(y, subgrid_size, image_size);
            float n = compute_n(l, m);
            float phase_offset = u_offset*l + v_offset*m + w_offset*n;
            lmnp_[y][x] = make_float4(l, m, n, phase_offset);
        }
    }
} // end compute_lmnp


/*
    This method has two modes:
        UPDATE_MODE_DERIV:
            Compute sums for the current station by combining aterm_derivatives
            with the aterm for station2 and storing the result in sum_deriv
        UPDATE_MODE_ATERM:
            Compute sums for every visibility by combining the aterms
            for station1 and station 2 and storing the result in sum_aterm
*/
#define UPDATE_MODE_DERIV 0
#define UPDATE_MODE_ATERM 1
template<int current_nr_terms, int mode>
__device__ void update_sums(
    const int                         subgrid_size,
    const float                       image_size,
    const unsigned int                total_nr_timesteps,
    const unsigned int                nr_channels,
    const unsigned int                nr_stations,
    const unsigned int                nr_terms,
    const unsigned int                term_offset,
    const UVW*           __restrict__ uvw,
    const float2*        __restrict__ aterm,
    const int*           __restrict__ aterm_indices,
    const float2*        __restrict__ aterm_derivatives,
    const float*         __restrict__ wavenumbers,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sum_aterm,
          float2*        __restrict__ sum_deriv,
          float4                      lmnp_[MAX_SUBGRID_SIZE][MAX_SUBGRID_SIZE],
          float2                      pixels_[MAX_NR_TERMS][NR_POLARIZATIONS][MAX_SUBGRID_SIZE])
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned nr_threads = blockDim.x * blockDim.y;
    unsigned s          = blockIdx.x;
    unsigned nr_pixels  = subgrid_size * subgrid_size;

    if (mode == UPDATE_MODE_ATERM) {
        assert(current_nr_terms == 1);
    }

    // Load metadata for first subgrid
    const Metadata &m0 = metadata[0];

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset_global = m.time_index - m0.time_index;
    const unsigned int station1     = m.baseline.station1;
    const unsigned int station2     = m.baseline.station2;
    const unsigned int nr_timesteps = m.nr_timesteps;

    // Iterate timesteps
    int current_nr_timesteps = 0;
    for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps) {
        int aterm_idx = aterm_indices[time_offset_global + time_offset_local];

        // Determine number of timesteps to process
        current_nr_timesteps = 0;
        for (int time = time_offset_local; time < nr_timesteps; time++) {
            if (aterm_indices[time_offset_global + time] == aterm_idx) {
                current_nr_timesteps++;
            } else {
                break;
            }
        }

        // Iterate batch of visibilities from the same timeslot
        for (int i = tid; i < ALIGN(current_nr_timesteps * nr_channels, nr_threads); i += nr_threads) {
            int time = (i / nr_channels) + time_offset_local;
            int chan = (i % nr_channels);

            // Load UVW
            float u, v, w;
            if (time < nr_timesteps) {
                u = uvw[time_offset_global + time].u;
                v = uvw[time_offset_global + time].v;
                w = uvw[time_offset_global + time].w;
            }

            // Load wavenumber
            float wavenumber = wavenumbers[chan];

            // Accumulate sums in registers
            float2 sum_xx[current_nr_terms];
            float2 sum_xy[current_nr_terms];
            float2 sum_yx[current_nr_terms];
            float2 sum_yy[current_nr_terms];
            for (unsigned int term_nr = 0; term_nr < current_nr_terms; term_nr++) {
                sum_xx[term_nr] = make_float2(0, 0);
                sum_xy[term_nr] = make_float2(0, 0);
                sum_yx[term_nr] = make_float2(0, 0);
                sum_yy[term_nr] = make_float2(0, 0);
            }

            // Iterate all rows of the subgrid
            for (unsigned int y = 0; y < subgrid_size; y++) {
                __syncthreads();

                // Precompute data for one row
                for (unsigned j = tid; j < (subgrid_size*nr_terms); j += nr_threads) {
                    unsigned int term_nr = j / subgrid_size;
                    unsigned int x       = j % subgrid_size;

                    if (term_nr < nr_terms) {
                        // Compute shifted position in subgrid
                        unsigned int x_src = (x + (subgrid_size/2)) % subgrid_size;
                        unsigned int y_src = (y + (subgrid_size/2)) % subgrid_size;

                        // Load pixels
                        float2 pixel[NR_POLARIZATIONS];
                        for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            unsigned int pixel_idx = index_subgrid(subgrid_size, s, pol, y_src, x_src);
                            pixel[pol] = subgrid[pixel_idx];
                        }

                        // Load first aterm
                        float2 *aterm1;
                        if (mode == UPDATE_MODE_DERIV) {
                            size_t station1_idx = index_aterm(subgrid_size, nr_terms, aterm_idx, term_offset+term_nr, y, x);
                            aterm1 = (float2 *) &aterm_derivatives[station1_idx];
                        } else {
                            size_t station1_idx = index_aterm(subgrid_size, nr_stations, aterm_idx, station1, y, x);
                            aterm1 = (float2 *) &aterm[station1_idx];
                        }

                        // Load second aterm
                        size_t station2_idx = index_aterm(subgrid_size, nr_stations, aterm_idx, station2, y, x);
                        float2 *aterm2 = (float2 *) &aterm[station2_idx];

                        // Apply aterm
                        apply_aterm_calibrate(pixel, aterm1, aterm2);

                        // Store pixels in shared memory
                        for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            pixels_[term_nr][pol][x] = pixel[pol];
                        }
                    } // end if
                } // end for j (subgrid_size * terms)

                __syncthreads();

                // Iterate all columns of current row
                for (unsigned int x = 0; x < subgrid_size; x++) {

                    // Load l,m,n
                    float l = lmnp_[y][x].x;
                    float m = lmnp_[y][x].y;
                    float n = lmnp_[y][x].z;

                    // Load phase offset
                    float phase_offset = lmnp_[y][x].w;

                    // Compute phase index
                    float phase_index = u*l + v*m + w*n;

                    // Compute phasor
                    float  phase  = (phase_index * wavenumber) - phase_offset;
                    float2 phasor = make_float2(raw_cos(phase), raw_sin(phase));

                    // Iterate all terms
                    for (unsigned int term_nr = 0; term_nr < current_nr_terms; term_nr++) {

                            // Load pixels
                            float2 pixel_xx = pixels_[term_nr][0][x];
                            float2 pixel_xy = pixels_[term_nr][1][x];
                            float2 pixel_yx = pixels_[term_nr][2][x];
                            float2 pixel_yy = pixels_[term_nr][3][x];

                            // Update sums
                            sum_xx[term_nr].x += phasor.x * pixel_xx.x;
                            sum_xx[term_nr].y += phasor.x * pixel_xx.y;
                            sum_xx[term_nr].x -= phasor.y * pixel_xx.y;
                            sum_xx[term_nr].y += phasor.y * pixel_xx.x;

                            sum_xy[term_nr].x += phasor.x * pixel_xy.x;
                            sum_xy[term_nr].y += phasor.x * pixel_xy.y;
                            sum_xy[term_nr].x -= phasor.y * pixel_xy.y;
                            sum_xy[term_nr].y += phasor.y * pixel_xy.x;

                            sum_yx[term_nr].x += phasor.x * pixel_yx.x;
                            sum_yx[term_nr].y += phasor.x * pixel_yx.y;
                            sum_yx[term_nr].x -= phasor.y * pixel_yx.y;
                            sum_yx[term_nr].y += phasor.y * pixel_yx.x;

                            sum_yy[term_nr].x += phasor.x * pixel_yy.x;
                            sum_yy[term_nr].y += phasor.x * pixel_yy.y;
                            sum_yy[term_nr].x -= phasor.y * pixel_yy.y;
                            sum_yy[term_nr].y += phasor.y * pixel_yy.x;
                    } // end for term_nr
                } // end for x
            } // end for y

            // Scale sums and store in device memory
            for (unsigned int term_nr = 0; term_nr < current_nr_terms; term_nr++) {
                const float scale = 1.0f / nr_pixels;
                if (time < nr_timesteps) {
                    unsigned int time_idx = time_offset_global + time;
                    unsigned int chan_idx = chan;
                    if (mode == UPDATE_MODE_DERIV) {
                        unsigned int sum_idx_xx = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, 0, time_idx, chan_idx);
                        unsigned int sum_idx_xy = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, 1, time_idx, chan_idx);
                        unsigned int sum_idx_yx = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, 2, time_idx, chan_idx);
                        unsigned int sum_idx_yy = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, 3, time_idx, chan_idx);
                        sum_deriv[sum_idx_xx] = sum_xx[term_nr] * scale;
                        sum_deriv[sum_idx_xy] = sum_xy[term_nr] * scale;
                        sum_deriv[sum_idx_yx] = sum_yx[term_nr] * scale;
                        sum_deriv[sum_idx_yy] = sum_yy[term_nr] * scale;
                    } else {
                        unsigned int sum_idx_xx = index_sum_aterm(total_nr_timesteps, nr_channels, 0, time_idx, chan_idx);
                        unsigned int sum_idx_xy = index_sum_aterm(total_nr_timesteps, nr_channels, 1, time_idx, chan_idx);
                        unsigned int sum_idx_yx = index_sum_aterm(total_nr_timesteps, nr_channels, 2, time_idx, chan_idx);
                        unsigned int sum_idx_yy = index_sum_aterm(total_nr_timesteps, nr_channels, 3, time_idx, chan_idx);
                        sum_aterm[sum_idx_xx] = sum_xx[term_nr] * scale;
                        sum_aterm[sum_idx_xy] = sum_xy[term_nr] * scale;
                        sum_aterm[sum_idx_yx] = sum_yx[term_nr] * scale;
                        sum_aterm[sum_idx_yy] = sum_yy[term_nr] * scale;
                    }
                }
            } // end for term_nr

            __syncthreads();

        } // end for i (visibilities)
    } // end for time_offset_local
} // end update_sums


__device__ void update_gradient(
    const int                         subgrid_size,
    const float                       image_size,
    const unsigned int                total_nr_timesteps,
    const unsigned int                nr_channels,
    const unsigned int                nr_stations,
    const unsigned int                nr_terms,
    const UVW*           __restrict__ uvw,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sum_aterm,
          float2*        __restrict__ sum_deriv,
          float2*        __restrict__ gradient,
          float4                      lmnp_[MAX_SUBGRID_SIZE][MAX_SUBGRID_SIZE])
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned nr_threads = blockDim.x * blockDim.y;
    unsigned s          = blockIdx.x;

    // Load metadata for first subgrid
    const Metadata &m0 = metadata[0];

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset  = m.time_index - m0.time_index;
    const unsigned int nr_timesteps = m.nr_timesteps;

    // Shared memory
    __shared__ float2 residual_[NR_POLARIZATIONS][MAX_NR_THREADS];
    __shared__ float2 gradient_[MAX_NR_TERMS];

    // Reset shared memory
    for (unsigned int i = tid; i < MAX_NR_TERMS; i += nr_threads) {
        if (i < MAX_NR_TERMS) {
            gradient_[i] = make_float2(0, 0);
        }
    }

    // Iterate all visibilities
    for (unsigned int i = tid; i < ALIGN(nr_timesteps*nr_channels, nr_threads); i += nr_threads) {
        unsigned int time = i / nr_channels;
        unsigned int chan = i % nr_channels;

        // Compute residual visibilities
        if (time < nr_timesteps) {
            unsigned int time_idx = time_offset + time;
            unsigned int chan_idx = chan;
            for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                unsigned int vis_idx = index_visibility(nr_channels, time_idx, chan_idx, pol);
                unsigned int sum_idx = index_sum_aterm(total_nr_timesteps, nr_channels, pol, time_idx, chan_idx);
                residual_[pol][tid] = (visibilities[vis_idx] - sum_aterm[sum_idx]) * weights[vis_idx];
            }
        } else {
            for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                residual_[pol][tid] = make_float2(0, 0);
            }
        }

        __syncthreads();

        // Iterate all terms
        for (unsigned term_nr = tid; term_nr < nr_terms; term_nr += nr_threads) {

            // Compute gradient update
            float2 update = make_float2(0, 0);

            // Iterate current batch of residuals
            for (unsigned j = 0; j < MAX_NR_THREADS; j++) {
                // Compute the index of the first visibility in the batch
                unsigned int k = i - tid + j;

                // Derive the current time and channel
                unsigned int time = k / nr_channels;
                unsigned int chan = k % nr_channels;

                if (term_nr < nr_terms && time < nr_timesteps) {
                    for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                        unsigned int time_idx = time_offset + time;
                        unsigned int chan_idx = chan;
                        unsigned int sum_idx = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr, pol, time_idx, chan_idx);
                        float2 sum      = sum_deriv[sum_idx];
                        float2 residual = residual_[pol][j];

                        if (term_nr < nr_terms) {
                            update.x += sum.x * residual.x;
                            update.x += sum.y * residual.y;
                            update.y += sum.x * residual.y;
                            update.y -= sum.y * residual.x;
                        }
                    } // end for pol
                } // end if
            } // end for threads

            // Update local gradient
            gradient_[term_nr] += update;
        } // end for term_nr

        __syncthreads();

    } // end for i (visibilities)

    // Iterate all terms * terms
    for (unsigned int i = tid; i < nr_terms; i += nr_threads) {
        if (i < nr_terms) {
            atomicAdd(&gradient[i], gradient_[i]);
        }
    }
} // end update_gradient


__device__ void update_hessian(
    const unsigned int                total_nr_timesteps,
    const unsigned int                nr_channels,
    const unsigned int                nr_stations,
    const unsigned int                nr_terms,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const Metadata*      __restrict__ metadata,
          float2*        __restrict__ sum_aterm,
          float2*        __restrict__ sum_deriv,
          float2*        __restrict__ hessian)
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned s          = blockIdx.x;
    unsigned nr_threads = blockDim.x * blockDim.y;

    // Metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset  = m.time_index;
    const unsigned int nr_timesteps = m.nr_timesteps;

    // Iterate all terms * terms
    for (unsigned int term_nr = tid; term_nr < (nr_terms*nr_terms); term_nr += nr_threads) {
        unsigned term_nr0 = term_nr / nr_terms;
        unsigned term_nr1 = term_nr % nr_terms;

        // Compute hessian update
        float2 update = make_float2(0, 0);

        // Iterate all timesteps
        for (unsigned int time = 0; time < nr_timesteps; time++) {

            // Iterate all channels
            for (unsigned int chan = 0; chan < nr_channels; chan++) {

                // Iterate all polarizations
                for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                    unsigned int time_idx = time_offset + time;
                    unsigned int chan_idx = chan;
                    unsigned int  vis_idx = index_visibility(nr_channels, time_idx, chan_idx, pol);
                    unsigned int sum_idx0 = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr0, pol, time_idx, chan_idx);
                    unsigned int sum_idx1 = index_sum_deriv(total_nr_timesteps, nr_channels, term_nr1, pol, time_idx, chan_idx);
                    float2 sum0 = sum_deriv[sum_idx0];
                    float2 sum1 = sum_deriv[sum_idx1] * weights[vis_idx];

                    // Update hessian
                    if (term_nr0 < nr_terms) {
                        update.x += sum0.x * sum1.x;
                        update.x += sum0.y * sum1.y;
                        update.y += sum0.y * sum1.x;
                        update.y -= sum0.x * sum1.y;
                    }
                } // end for pol
            } // end chan
        } // end for time

        // Update local hessian
        if (term_nr0 < nr_terms) {
            atomicAdd(&hessian[term_nr], update);
        }
    } // end for term_nr (terms * terms)
} // end update_hessian


#define UPDATE_SUMS(current_nr_terms, mode) \
    for (; (term_offset + current_nr_terms) <= nr_terms; term_offset += current_nr_terms) { \
        update_sums<current_nr_terms, mode>( \
                subgrid_size, image_size, total_nr_timesteps, nr_channels, nr_stations, \
                nr_terms, term_offset, \
                uvw, aterm, aterm_indices, aterm_derivatives, wavenumbers, metadata, subgrid, \
                sum_aterm, sum_deriv, lmnp_, pixels_); \
    }

extern "C" {

__global__ void kernel_calibrate_sums(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         total_nr_timesteps,
    const int                         nr_channels,
    const int                         nr_stations,
    const int                         nr_terms,
    const UVW*           __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const int*           __restrict__ aterm_indices,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sum_aterm,
          float2*        __restrict__ sum_deriv,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    // Shared memory
    __shared__ float4 lmnp_[MAX_SUBGRID_SIZE][MAX_SUBGRID_SIZE];
    __shared__ float2 pixels_[MAX_NR_TERMS][NR_POLARIZATIONS][MAX_SUBGRID_SIZE];

    compute_lmnp(grid_size, subgrid_size, image_size, w_step, metadata, lmnp_);

    int term_offset;

    // Compute sum_deriv
    term_offset = 0;
    UPDATE_SUMS(8, UPDATE_MODE_DERIV)
    UPDATE_SUMS(7, UPDATE_MODE_DERIV)
    UPDATE_SUMS(6, UPDATE_MODE_DERIV)
    UPDATE_SUMS(5, UPDATE_MODE_DERIV)
    UPDATE_SUMS(4, UPDATE_MODE_DERIV)
    UPDATE_SUMS(3, UPDATE_MODE_DERIV)
    UPDATE_SUMS(2, UPDATE_MODE_DERIV)
    UPDATE_SUMS(1, UPDATE_MODE_DERIV)

    // Compute sum_aterm
    term_offset = 0;
    UPDATE_SUMS(1, UPDATE_MODE_ATERM)
} // end kernel_calibrate_sums


__global__ void kernel_calibrate_gradient(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         total_nr_timesteps,
    const int                         nr_channels,
    const int                         nr_stations,
    const int                         nr_terms,
    const UVW*           __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const int*           __restrict__ aterm_indices,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sum_aterm,
          float2*        __restrict__ sum_deriv,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    __shared__ float4 lmnp_[MAX_SUBGRID_SIZE][MAX_SUBGRID_SIZE];

    compute_lmnp(grid_size, subgrid_size, image_size, w_step, metadata, lmnp_);

    update_gradient(
        subgrid_size, image_size, total_nr_timesteps,
        nr_channels, nr_stations, nr_terms,
        uvw, aterm, aterm_derivatives,
        wavenumbers, visibilities, weights, metadata, subgrid,
        sum_aterm, sum_deriv, gradient, lmnp_);
} // end kernel_calibrate_gradient


__global__ void kernel_calibrate_hessian(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         total_nr_timesteps,
    const int                         nr_channels,
    const int                         nr_stations,
    const int                         nr_terms,
    const UVW*           __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const int*           __restrict__ aterm_indices,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sum_aterm,
          float2*        __restrict__ sum_deriv,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    update_hessian(
        total_nr_timesteps, nr_channels, nr_stations, nr_terms,
        visibilities, weights, metadata, sum_aterm, sum_deriv, hessian);
} // end kernel_calibrate_hessian

} // end extern "C"
