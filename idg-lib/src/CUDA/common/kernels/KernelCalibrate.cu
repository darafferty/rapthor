#include "hip/hip_runtime.h"
#include "Types.h"
#include "math.cu"

#define ALIGN(N,A) (((N)+(A)-1)/(A)*(A))
#define MAX_NR_TERMS     8
#define BATCH_SIZE_PIXELS     64

inline __device__ long index_sums(
    unsigned int total_nr_timesteps, // number of timesteps for all baselines
    unsigned int nr_channels,        // number channels for a single baseline
    unsigned int term_nr,
    unsigned int pol,
    unsigned int time,
    unsigned int chan)
{
    // sums: [MAX_NR_TERMS][NR_POLARIZATIONS][TOTAL_NR_TIMESTEPS][NR_CHANNELS]
    return term_nr * NR_POLARIZATIONS * total_nr_timesteps * nr_channels +
           pol * total_nr_timesteps * nr_channels +
           time * nr_channels +
           chan;
}

inline __device__ long index_lmnp(
        unsigned int subgrid_size,
        unsigned int s,
        unsigned int y,
        unsigned int x)
{
    // lmnp: [NR_SUBGRIDS][SUBGRIDSIZE][SUBGRIDSIZE]
    return s * subgrid_size * subgrid_size +
           y * subgrid_size + x;
}

extern "C" {

__global__ void kernel_calibrate_lmnp(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         total_nr_timesteps,
    const int                         nr_channels,
    const int                         nr_stations,
    const int                         nr_terms,
    const UVW<float>*    __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const int*           __restrict__ aterm_indices,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sums,
          float4*        __restrict__ lmnp,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned nr_threads = blockDim.x * blockDim.y;
    unsigned s          = blockIdx.x;
    unsigned nr_pixels  = subgrid_size * subgrid_size;

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const Coordinate coordinate = m.coordinate;

    // Location of current subgrid
    const int x_coordinate = coordinate.x;
    const int y_coordinate = coordinate.y;
    const int z_coordinate = coordinate.z;

    for (unsigned int i = tid; i < nr_pixels; i += nr_threads) {
        unsigned int y = i / subgrid_size;
        unsigned int x = i % subgrid_size;

        if (y < subgrid_size) {

            // Compute u,v,w offset in wavelenghts
            const float u_offset = (x_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
            const float v_offset = (y_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
            const float w_offset = w_step * ((float) z_coordinate + 0.5) * 2 * M_PI;

            // Compute l,m,n and phase_offset
            float l = compute_l(x, subgrid_size, image_size);
            float m = compute_m(y, subgrid_size, image_size);
            float n = compute_n(l, m);
            float phase_offset = u_offset*l + v_offset*m + w_offset*n;

            // Store result
            unsigned int lmnp_idx = index_lmnp(subgrid_size, s, y, x);
            lmnp[lmnp_idx] = make_float4(l, m, n, phase_offset);
        }
    } // end for i
} // end kernel_calibrate_lmnp


__global__ void kernel_calibrate_sums(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         total_nr_timesteps,
    const int                         nr_channels,
    const int                         nr_stations,
    const int                         nr_terms,
    const UVW<float>*    __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const int*           __restrict__ aterm_indices,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sums,
          float4*        __restrict__ lmnp,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned nr_threads = blockDim.x * blockDim.y;
    unsigned s          = blockIdx.x;
    unsigned nr_pixels  = subgrid_size * subgrid_size;

    // Load metadata for first subgrid
    const Metadata &m0 = metadata[0];

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset_global = m.time_index - m0.time_index;
    const unsigned int station2     = m.baseline.station2;
    const unsigned int nr_timesteps = m.nr_timesteps;

    // Shared memory
    __shared__ float2 pixels_[MAX_NR_TERMS][NR_POLARIZATIONS][BATCH_SIZE_PIXELS];
    __shared__ float4 lmnp_[BATCH_SIZE_PIXELS];

    // Iterate timesteps
    int current_nr_timesteps = 0;
    for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps) {
        int aterm_idx = aterm_indices[time_offset_global + time_offset_local];

        // Determine number of timesteps to process
        current_nr_timesteps = 0;
        for (int time = time_offset_local; time < nr_timesteps; time++) {
            if (aterm_indices[time_offset_global + time] == aterm_idx) {
                current_nr_timesteps++;
            } else {
                break;
            }
        }

        // Iterate batch of visibilities from the same timeslot
        for (int i = tid; i < ALIGN(current_nr_timesteps * nr_channels, nr_threads); i += nr_threads) {
            unsigned int time_idx_batch  = (i / nr_channels);
            unsigned int chan_idx_local  = (i % nr_channels);
            unsigned int time_idx_local  = time_offset_local + time_idx_batch;
            unsigned int time_idx_global = time_offset_global + time_idx_local;

            // Load UVW
            float u, v, w;
            if (time_idx_batch < current_nr_timesteps) {
                u = uvw[time_idx_global].u;
                v = uvw[time_idx_global].v;
                w = uvw[time_idx_global].w;
            }

            // Load wavenumber
            float wavenumber = wavenumbers[chan_idx_local];


            // Accumulate sums in registers
            float2 sum[MAX_NR_TERMS][NR_POLARIZATIONS] = {0, 0};

            // Iterate all pixels
            for (unsigned int pixel_offset = 0; pixel_offset < nr_pixels; pixel_offset += BATCH_SIZE_PIXELS) {
                __syncthreads();

                for (unsigned int j = tid; j < BATCH_SIZE_PIXELS; j += nr_threads) {
                    unsigned int y = (pixel_offset + j) / subgrid_size;
                    unsigned int x = (pixel_offset + j) % subgrid_size;

                    for (unsigned int term_nr = 0; term_nr < MAX_NR_TERMS; term_nr++) {
                        // Reset pixel to zero
                        for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            pixels_[term_nr][pol][j] = make_float2(0, 0);
                        }

                        // Prepare batch
                        if (y < subgrid_size) {
                            // Compute shifted position in subgrid
                            unsigned int x_src = (x + (subgrid_size/2)) % subgrid_size;
                            unsigned int y_src = (y + (subgrid_size/2)) % subgrid_size;

                             // Load pixel and aterms
                            float2 pixel[NR_POLARIZATIONS];
                            float2 aterm1[NR_POLARIZATIONS];
                            float2 aterm2[NR_POLARIZATIONS];
                            for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                                unsigned int term_idx   = term_nr;
                                unsigned int pixel_idx  = index_subgrid(subgrid_size, s, pol, y_src, x_src);
                                unsigned int aterm1_idx = index_aterm_transposed(subgrid_size, nr_terms, aterm_idx, term_idx, y, x, pol);
                                unsigned int aterm2_idx = index_aterm_transposed(subgrid_size, nr_stations, aterm_idx, station2, y, x, pol);
                                pixel[pol]  = subgrid[pixel_idx];
                                aterm1[pol] = aterm_derivatives[aterm1_idx];
                                aterm2[pol] = aterm[aterm2_idx];
                            }

                            // Apply aterm
                            apply_aterm_calibrate(pixel, aterm1, aterm2);

                            // Store pixel in shared memory
                            for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                                pixels_[term_nr][pol][j] = pixel[pol];
                            }

                        } // end if
                    } // end for term_nr

                    // Load l,m,n and phase_offset into shared memory
                    if (y < subgrid_size) {
                        unsigned int lmnp_idx = index_lmnp(subgrid_size, s, y, x);
                        lmnp_[j] = lmnp[lmnp_idx];
                    }
                } // end for j

                __syncthreads();

                // Iterate batch
                for (unsigned int j = 0; j < BATCH_SIZE_PIXELS; j++) {
                    // Load l,m,n and phase_offset
                    float l = lmnp_[j].x;
                    float m = lmnp_[j].y;
                    float n = lmnp_[j].z;
                    float phase_offset = lmnp_[j].w;

                    // Compute phase index
                    float phase_index = u*l + v*m + w*n;

                    // Compute phasor
                    float  phase  = (phase_index * wavenumber) - phase_offset;
                    float2 phasor = make_float2(raw_cos(phase), raw_sin(phase));

                    // Update sum
                    for (unsigned int term_nr = 0; term_nr < MAX_NR_TERMS; term_nr++) {
                        for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            sum[term_nr][pol] += (phasor * pixels_[term_nr][pol][j]);
                        } // end for pol
                    } // end for term_nr
                } // end for j (batch)
            } // end for pixel_offset

            const float scale = 1.0f / nr_pixels;
            if (time_idx_batch < current_nr_timesteps) {
                for (unsigned int term_nr = 0; term_nr < MAX_NR_TERMS; term_nr++) {
                    for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                        unsigned int sum_idx = index_sums(total_nr_timesteps, nr_channels, term_nr, pol, time_idx_global, chan_idx_local);
                        sums[sum_idx] = conj(sum[term_nr][pol]) * scale;
                    } // end for pol
                } // end for term_nr
            } // end if time

        } // end for i (visibilities)
    } // end for time_offset_local
} // end kernel_calibrate_sums


__global__ void kernel_calibrate_gradient(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         total_nr_timesteps,
    const int                         nr_channels,
    const int                         nr_stations,
    const int                         nr_terms,
    const UVW<float>*    __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const int*           __restrict__ aterm_indices,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sums,
          float4*        __restrict__ lmnp,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned nr_threads = blockDim.x * blockDim.y;
    unsigned s          = blockIdx.x;
    unsigned nr_pixels  = subgrid_size * subgrid_size;

    // Load metadata for first subgrid
    const Metadata &m0 = metadata[0];

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset_global = m.time_index - m0.time_index;
    const unsigned int station1     = m.baseline.station1;
    const unsigned int station2     = m.baseline.station2;
    const unsigned int nr_timesteps = m.nr_timesteps;

    // Shared memory
    __shared__ float2 pixels_[NR_POLARIZATIONS][BATCH_SIZE_PIXELS];
    __shared__ float4 lmnp_[BATCH_SIZE_PIXELS];

    // Accumulate gradient update in registers
    float2 update[MAX_NR_TERMS];

    // Iterate timesteps
    int current_nr_timesteps = 0;
    for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps) {
        int aterm_idx = aterm_indices[time_offset_global + time_offset_local];

        // Reset update to zero
        for (unsigned int term_nr = 0; term_nr < MAX_NR_TERMS; term_nr++) {
            update[term_nr] = make_float2(0, 0);
        }

        // Determine number of timesteps to process
        current_nr_timesteps = 0;
        for (int time = time_offset_local; time < nr_timesteps; time++) {
            if (aterm_indices[time_offset_global + time] == aterm_idx) {
                current_nr_timesteps++;
            } else {
                break;
            }
        }

        // Iterate batch of visibilities from the same timeslot
        for (int i = tid; i < ALIGN(current_nr_timesteps * nr_channels, nr_threads); i += nr_threads) {
            unsigned int time_idx_batch  = (i / nr_channels);
            unsigned int chan_idx_local  = (i % nr_channels);
            unsigned int time_idx_local  = time_offset_local + time_idx_batch;
            unsigned int time_idx_global = time_offset_global + time_idx_local;

            // Load UVW
            float u, v, w;
            if (time_idx_batch < current_nr_timesteps) {
                u = uvw[time_idx_global].u;
                v = uvw[time_idx_global].v;
                w = uvw[time_idx_global].w;
            }

            // Load wavenumber
            float wavenumber = wavenumbers[chan_idx_local];

            // Accumulate sums in registers
            float2 sum[NR_POLARIZATIONS] = {0, 0};

            // Iterate all pixels
            for (unsigned int pixel_offset = 0; pixel_offset < nr_pixels; pixel_offset += BATCH_SIZE_PIXELS) {
                __syncthreads();

                for (unsigned int j = tid; j < BATCH_SIZE_PIXELS; j += nr_threads) {
                    unsigned int y = (pixel_offset + j) / subgrid_size;
                    unsigned int x = (pixel_offset + j) % subgrid_size;

                    // Reset pixel to zero
                    for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                        pixels_[pol][j] = make_float2(0, 0);
                    }

                    // Prepare batch
                    if (y < subgrid_size) {
                        // Compute shifted position in subgrid
                        unsigned int x_src = (x + (subgrid_size/2)) % subgrid_size;
                        unsigned int y_src = (y + (subgrid_size/2)) % subgrid_size;

                        // Load pixel and aterms
                        float2 pixel[NR_POLARIZATIONS];
                        float2 aterm1[NR_POLARIZATIONS];
                        float2 aterm2[NR_POLARIZATIONS];
                        for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            unsigned int pixel_idx  = index_subgrid(subgrid_size, s, pol, y_src, x_src);
                            unsigned int aterm1_idx = index_aterm_transposed(subgrid_size, nr_stations, aterm_idx, station1, y, x, pol);
                            unsigned int aterm2_idx = index_aterm_transposed(subgrid_size, nr_stations, aterm_idx, station2, y, x, pol);
                            pixel[pol]  = subgrid[pixel_idx];
                            aterm1[pol] = aterm[aterm1_idx];
                            aterm2[pol] = aterm[aterm2_idx];
                        }

                        // Apply aterm
                        apply_aterm_calibrate(pixel, aterm1, aterm2);

                        // Store pixel in shared memory
                        for (unsigned pol = 0; pol < NR_POLARIZATIONS; pol++) {
                            pixels_[pol][j] = pixel[pol];
                        }

                        // Load l,m,n and phase_offset into shared memory
                        unsigned int lmnp_idx = index_lmnp(subgrid_size, s, y, x);
                        lmnp_[j] = lmnp[lmnp_idx];
                    } // end if
                } // end for j

                __syncthreads();

                // Iterate batch
                for (unsigned int j = 0; j < BATCH_SIZE_PIXELS; j++) {
                    // Load l,m,n and phase_offset
                    float l = lmnp_[j].x;
                    float m = lmnp_[j].y;
                    float n = lmnp_[j].z;
                    float phase_offset = lmnp_[j].w;

                    // Compute phase index
                    float phase_index = u*l + v*m + w*n;

                    // Compute phasor
                    float  phase  = (phase_index * wavenumber) - phase_offset;
                    float2 phasor = make_float2(raw_cos(phase), raw_sin(phase));

                    // Update sum
                    for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                        sum[pol] += phasor * pixels_[pol][j];
                    }
                } // end for j (batch)
            } // end for pixel_offset

            if (time_idx_batch < current_nr_timesteps) {

                // Compute residual
                float2 residual[NR_POLARIZATIONS];
                const float scale = 1.0f / nr_pixels;
                for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                    unsigned int vis_idx = index_visibility(nr_channels, time_idx_global, chan_idx_local, pol);
                    residual[pol] = (visibilities[vis_idx] - (sum[pol] * scale)) * weights[vis_idx];

                    // Compute gradient update
                    for (unsigned int term_nr = 0; term_nr < MAX_NR_TERMS; term_nr++) {
                        unsigned int sum_idx = index_sums(total_nr_timesteps, nr_channels, term_nr, pol, time_idx_global, chan_idx_local);
                        update[term_nr] += residual[pol] * sums[sum_idx];
                    } // end for term
                } // end for pol
            } // end if time

            __syncthreads();

        } // end for i (visibilities)

        // Update gradient
        for (unsigned int term_nr = 0; term_nr < nr_terms; term_nr++) {
            unsigned int idx = aterm_idx * nr_terms + term_nr;
            atomicAdd(&gradient[idx], update[term_nr]);
        }
    } // end for time_offset_local
} // end kernel_calibrate_gradient


__global__ void kernel_calibrate_hessian(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         total_nr_timesteps,
    const int                         nr_channels,
    const int                         nr_stations,
    const int                         nr_terms,
    const UVW<float>*    __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
    const float2*        __restrict__ visibilities,
    const float*         __restrict__ weights,
    const float2*        __restrict__ aterm,
    const float2*        __restrict__ aterm_derivatives,
    const int*           __restrict__ aterm_indices,
    const Metadata*      __restrict__ metadata,
    const float2*        __restrict__ subgrid,
          float2*        __restrict__ sums,
          float4*        __restrict__ lmnp,
          float2*        __restrict__ hessian,
          float2*        __restrict__ gradient)
{
    unsigned tidx       = threadIdx.x;
    unsigned tidy       = threadIdx.y;
    unsigned tid        = tidx + tidy * blockDim.x;
    unsigned s          = blockIdx.x;
    unsigned nr_threads = blockDim.x * blockDim.y;

    // Metadata for current subgrid
    const Metadata &m = metadata[s];
    const unsigned int time_offset_global = m.time_index;
    const unsigned int nr_timesteps = m.nr_timesteps;

    // Iterate timesteps
    int current_nr_timesteps = 0;
    for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps) {
        int aterm_idx = aterm_indices[time_offset_global + time_offset_local];

        // Determine number of timesteps to process
        current_nr_timesteps = 0;
        for (int time = time_offset_local; time < nr_timesteps; time++) {
            if (aterm_indices[time_offset_global + time] == aterm_idx) {
                current_nr_timesteps++;
            } else {
                break;
            }
        }

        // Iterate all terms * terms
        for (unsigned int term_nr = tid; term_nr < (nr_terms*nr_terms); term_nr += nr_threads) {
            unsigned term_nr1 = term_nr / nr_terms;
            unsigned term_nr0 = term_nr % nr_terms;

            // Compute hessian update
            float2 update = make_float2(0, 0);

            // Iterate all timesteps
            for (unsigned int time = 0; time < current_nr_timesteps; time++) {

                // Iterate all channels
                for (unsigned int chan = 0; chan < nr_channels; chan++) {

                    // Iterate all polarizations
                    for (unsigned int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                        unsigned int time_idx_global = time_offset_global + time_offset_local + time;
                        unsigned int chan_idx = chan;
                        unsigned int  vis_idx = index_visibility(nr_channels, time_idx_global, chan_idx, pol);
                        unsigned int sum_idx0 = index_sums(total_nr_timesteps, nr_channels, term_nr0, pol, time_idx_global, chan_idx);
                        unsigned int sum_idx1 = index_sums(total_nr_timesteps, nr_channels, term_nr1, pol, time_idx_global, chan_idx);
                        float2 sum0 = sums[sum_idx0];
                        float2 sum = conj(sums[sum_idx1]) * weights[vis_idx];

                        // Update hessian
                        if (term_nr0 < nr_terms) {
                            update += sum0 * sum;
                        }
                    } // end for pol
                } // end chan
            } // end for time

            __syncthreads();

            // Update hessian
            if (term_nr0 < nr_terms) {
                unsigned idx = aterm_idx * nr_terms * nr_terms + term_nr1 * nr_terms + term_nr0;
                atomicAdd(&hessian[idx], update);
            }
        } // end for term_nr (terms * terms)
    } // end for time_offset_local
} // end kernel_calibrate_hessian

} // end extern "C"
