#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>

#include "Types.h"
#include "math.cu"

#define BATCH_SIZE GRIDDER_BATCH_SIZE

/*
	Kernel
*/
template<int current_nr_channels>
__device__ void kernel_gridder_(
    const int gridsize,
    const float imagesize,
    const float w_offset,
    const int nr_channels,
    const int channel_offset,
    const int nr_stations,
	const UVWType			__restrict__ uvw,
	const WavenumberType	__restrict__ wavenumbers,
	const VisibilitiesType	__restrict__ visibilities,
	const SpheroidalType	__restrict__ spheroidal,
	const ATermType			__restrict__ aterm,
	const MetadataType		__restrict__ metadata,
	SubGridType				__restrict__ subgrid
	) {
	int tidx = threadIdx.x;
	int tidy = threadIdx.y;
	int tid = tidx + tidy * blockDim.x;
	int blockSize = blockDim.x * blockDim.y;
    int s = blockIdx.x;

    // Set subgrid to zero
    if (channel_offset == 0) {
        for (int i = tid; i < SUBGRIDSIZE * SUBGRIDSIZE; i += blockSize) {
            subgrid[s][0][0][i] = make_float2(0, 0);
            subgrid[s][1][0][i] = make_float2(0, 0);
            subgrid[s][2][0][i] = make_float2(0, 0);
            subgrid[s][3][0][i] = make_float2(0, 0);
        }
    }

    __syncthreads();

    // Load metadata for first subgrid
    const Metadata &m_0 = metadata[0];

    // Load metadata for current subgrid
	const Metadata &m = metadata[s];
    const int time_offset_global = (m.baseline_offset - m_0.baseline_offset) + (m.time_offset - m_0.time_offset);
    const int nr_timesteps = m.nr_timesteps;
	const int aterm_index = m.aterm_index;
	const int station1 = m.baseline.station1;
	const int station2 = m.baseline.station2;
	const int x_coordinate = m.coordinate.x;
	const int y_coordinate = m.coordinate.y;

    // Shared data
	__shared__ float4 _visibilities[2][BATCH_SIZE][current_nr_channels];
	__shared__ float4 _uvw[BATCH_SIZE];
	__shared__ float _wavenumbers[current_nr_channels];

    // Load wavenumbers
    for (int i = tid; i < current_nr_channels; i += blockSize) {
        _wavenumbers[i] = wavenumbers[channel_offset + i];
    }

    // Iterate all timesteps
    int current_nr_timesteps = BATCH_SIZE;
    for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps) {
        current_nr_timesteps = nr_timesteps - time_offset_local < BATCH_SIZE ?
                               nr_timesteps - time_offset_local : BATCH_SIZE;

        __syncthreads();

	    // Load UVW
	    for (int time = tid; time < current_nr_timesteps; time += blockSize) {
            UVW a = uvw[time_offset_global + time_offset_local + time];
            _uvw[time] = make_float4(a.u, a.v, a.w, 0);
        }

	    // Load visibilities
	    for (int i = tid; i < current_nr_timesteps * current_nr_channels; i += blockSize) {
            int time = i / current_nr_channels;
            int chan = i % current_nr_channels;
            int index = (time_offset_global + time_offset_local + time) * nr_channels + (channel_offset + chan);
            float2 a = visibilities[index][0];
            float2 b = visibilities[index][1];
            float2 c = visibilities[index][2];
            float2 d = visibilities[index][3];
            _visibilities[0][time][chan] = make_float4(a.x, a.y, b.x, b.y);
            _visibilities[1][time][chan] = make_float4(c.x, c.y, d.x, d.y);
        }

	    __syncthreads();

        // Compute u and v offset in wavelenghts
        float u_offset = (x_coordinate + SUBGRIDSIZE/2 - gridsize/2) / imagesize * 2 * M_PI;
        float v_offset = (y_coordinate + SUBGRIDSIZE/2 - gridsize/2) / imagesize * 2 * M_PI;

	    // Iterate all pixels in subgrid
        for (int i = tid; i < SUBGRIDSIZE * SUBGRIDSIZE; i += blockSize) {
            int y = i / SUBGRIDSIZE;
            int x = i % SUBGRIDSIZE;

            // Private pixels
            float2 uvXX = make_float2(0, 0);
            float2 uvXY = make_float2(0, 0);
            float2 uvYX = make_float2(0, 0);
            float2 uvYY = make_float2(0, 0);

            // Compute l,m,n
            float l = (x+0.5-(SUBGRIDSIZE/2)) * imagesize/SUBGRIDSIZE;
            float m = (y+0.5-(SUBGRIDSIZE/2)) * imagesize/SUBGRIDSIZE;
            float n = 1.0f - (float) sqrt(1.0 - (double) (l * l) - (double) (m * m));

            // Iterate all timesteps
            for (int time = 0; time < current_nr_timesteps; time++) {
                // Load UVW coordinates
                float u = _uvw[time].x;
                float v = _uvw[time].y;
                float w = _uvw[time].z;

                // Compute phase index
                float phase_index = u*l + v*m + w*n;

                // Compute phase offset
                float phase_offset = u_offset*l + v_offset*m + w_offset*n;

                // Compute phasor
                #pragma unroll current_nr_channels
                for (int chan = 0; chan < current_nr_channels; chan++) {
                    float wavenumber = _wavenumbers[chan];
                    float phase = phase_offset - (phase_index * wavenumber);
                    float2 phasor = make_float2(cos(phase), sin(phase));

                    // Load visibilities from shared memory
                    float4 a = _visibilities[0][time][chan];
                    float4 b = _visibilities[1][time][chan];
                    float2 visXX = make_float2(a.x, a.y);
                    float2 visXY = make_float2(a.z, a.w);
                    float2 visYX = make_float2(b.x, b.y);
                    float2 visYY = make_float2(b.z, b.w);

                    // Multiply visibility by phasor
                    uvXX.x += phasor.x * visXX.x;
                    uvXX.y += phasor.x * visXX.y;
                    uvXX.x -= phasor.y * visXX.y;
                    uvXX.y += phasor.y * visXX.x;

                    uvXY.x += phasor.x * visXY.x;
                    uvXY.y += phasor.x * visXY.y;
                    uvXY.x -= phasor.y * visXY.y;
                    uvXY.y += phasor.y * visXY.x;

                    uvYX.x += phasor.x * visYX.x;
                    uvYX.y += phasor.x * visYX.y;
                    uvYX.x -= phasor.y * visYX.y;
                    uvYX.y += phasor.y * visYX.x;

                    uvYY.x += phasor.x * visYY.x;
                    uvYY.y += phasor.x * visYY.y;
                    uvYY.x -= phasor.y * visYY.y;
                    uvYY.y += phasor.y * visYY.x;
                }
            } // end for time

            // Get aterm for station1
            float2 aXX1 = aterm[aterm_index * nr_stations + station1][y][x][0];
            float2 aXY1 = aterm[aterm_index * nr_stations + station1][y][x][1];
            float2 aYX1 = aterm[aterm_index * nr_stations + station1][y][x][2];
            float2 aYY1 = aterm[aterm_index * nr_stations + station1][y][x][3];

            // Get aterm for station2
            float2 aXX2 = hipConjf(aterm[aterm_index * nr_stations + station2][y][x][0]);
            float2 aXY2 = hipConjf(aterm[aterm_index * nr_stations + station2][y][x][1]);
            float2 aYX2 = hipConjf(aterm[aterm_index * nr_stations + station2][y][x][2]);
            float2 aYY2 = hipConjf(aterm[aterm_index * nr_stations + station2][y][x][3]);

            // Apply aterm
            apply_aterm(
                aXX1, aXY1, aYX1, aYY1,
                aXX2, aXY2, aYX2, aYY2,
                uvXX, uvXY, uvYX, uvYY);

            // Load spheroidal
            float sph = spheroidal[y][x];

            // Compute shifted position in subgrid
            int x_dst = (x + (SUBGRIDSIZE/2)) % SUBGRIDSIZE;
            int y_dst = (y + (SUBGRIDSIZE/2)) % SUBGRIDSIZE;

            // Set subgrid value
            subgrid[s][0][y_dst][x_dst] += uvXX * sph;
            subgrid[s][1][y_dst][x_dst] += uvXY * sph;
            subgrid[s][2][y_dst][x_dst] += uvYX * sph;
            subgrid[s][3][y_dst][x_dst] += uvYY * sph;
	    } // end for i
    } // end for time_offset_local
}

extern "C" {
__global__ void kernel_gridder(
    const int gridsize,
    const float imagesize,
    const float w_offset,
    const int nr_channels,
    const int nr_stations,
	const UVWType			__restrict__ uvw,
	const WavenumberType	__restrict__ wavenumbers,
	const VisibilitiesType	__restrict__ visibilities,
	const SpheroidalType	__restrict__ spheroidal,
	const ATermType			__restrict__ aterm,
	const MetadataType		__restrict__ metadata,
	SubGridType				__restrict__ subgrid
	) {
    int channel_offset = 0;
    for (; (channel_offset + 8) <= nr_channels; channel_offset += 8) {
        kernel_gridder_<8>(
            gridsize, imagesize, w_offset, nr_channels, channel_offset, nr_stations,
            uvw, wavenumbers, visibilities, spheroidal, aterm, metadata, subgrid);
    }

    for (; (channel_offset + 4) <= nr_channels; channel_offset += 4) {
        kernel_gridder_<4>(
            gridsize, imagesize, w_offset, nr_channels, channel_offset, nr_stations,
            uvw, wavenumbers, visibilities, spheroidal, aterm, metadata, subgrid);
    }

    for (; (channel_offset + 2) <= nr_channels; channel_offset += 2) {
        kernel_gridder_<2>(
            gridsize, imagesize, w_offset, nr_channels, channel_offset, nr_stations,
            uvw, wavenumbers, visibilities, spheroidal, aterm, metadata, subgrid);
    }

    for (; channel_offset < nr_channels; channel_offset++) {
        kernel_gridder_<1>(
            gridsize, imagesize, w_offset, nr_channels, channel_offset, nr_stations,
            uvw, wavenumbers, visibilities, spheroidal, aterm, metadata, subgrid);
    }
}
}
