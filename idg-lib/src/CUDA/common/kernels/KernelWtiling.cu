#include "hip/hip_runtime.h"
// Copyright (C) 2021 ASTRON (Netherlands Institute for Radio Astronomy)
// SPDX-License-Identifier: GPL-3.0-or-later

#include <hip/hip_complex.h>

#include "Types.h"
#include "math.cu"

extern "C" {
__global__ void kernel_copy_tiles(
    const unsigned int             src_tile_size,
    const unsigned int             dst_tile_size,
    const int*        __restrict__ src_tile_ids,
    const int*        __restrict__ dst_tile_ids,
          float2*     __restrict__ src_tiles,
          float2*     __restrict__ dst_tiles)
{
    // Map blockIdx.x to polarizations
    assert(gridDim.x == NR_POLARIZATIONS);
    unsigned int pol = blockIdx.x;

    // Map blockIdx.y to tile_id
    unsigned int src_tile_index = src_tile_ids[blockIdx.y];
    unsigned int dst_tile_index = dst_tile_ids[blockIdx.y];

    // Map threadIdx.x to thread id
    unsigned int tid = threadIdx.x;

    // Compute the number of threads working on one polarizaton of a tile
    unsigned int nr_threads = blockDim.x;

    // Compute remaining parameters
    int padding = dst_tile_size - src_tile_size;
    int copy_tile_size = min(src_tile_size, dst_tile_size);

    // Reset dst_tile to zero if src_tile is smaller
    if (padding > 0)
    {
        for (unsigned int i = tid; i < (dst_tile_size * dst_tile_size); i += nr_threads)
        {
            unsigned int y = i / dst_tile_size;
            unsigned int x = i % dst_tile_size;

            if (y < dst_tile_size)
            {
                size_t dst_idx = index_grid(dst_tile_size, dst_tile_index, pol, y, x);
                dst_tiles[dst_idx] = make_float2(0, 0);
            }
        }
    }

    __syncthreads();

    // Copy src_tile to dst_tile and reset src_tile to zero
    for (unsigned int i = tid; i < (copy_tile_size * copy_tile_size); i += nr_threads)
    {
        unsigned int src_y = i / copy_tile_size;
        unsigned int src_x = i % copy_tile_size;
        unsigned int dst_y = src_y;
        unsigned int dst_x = src_x;

        if (padding > 0) {
            dst_y += padding / 2;
            dst_x += padding / 2;
        } else if (padding < 0) {
            src_y -= padding / 2;
            src_x -= padding / 2;
        }

        if (src_y < src_tile_size && dst_y < dst_tile_size)
        {
            size_t dst_idx = index_grid(dst_tile_size, dst_tile_index, pol, dst_y, dst_x);
            size_t src_idx = index_grid(src_tile_size, src_tile_index, pol, src_y, src_x);
            dst_tiles[dst_idx] = src_tiles[src_idx];
            src_tiles[src_idx] = make_float2(0, 0);
        }
    }
} // end kernel_copy_tiles

__global__ void kernel_apply_phasor(
    const float                    image_size,
    const float                    w_step,
    const int                      tile_size,
          float2*     __restrict__ tiles,
    const float*      __restrict__ shift,
    const Coordinate* __restrict__ tile_coordinates)
{
    // Map blockIdx.x to polarizations
    assert(gridDim.x == NR_POLARIZATIONS);
    unsigned int pol = blockIdx.x;

    // Map blockIdx.y to tile_index
    unsigned int tile_index = blockIdx.y;

    // Map threadIdx.x to thread id
    unsigned int tid = threadIdx.x;

    // Compute the number of threads working on one polarizaton of a tile
    unsigned int nr_threads = blockDim.x;

    // Compute cell_size
    float cell_size = image_size / tile_size;

    // Compute scale
    float scale = 1.0f / (tile_size * tile_size);

    // Compute W
    const Coordinate& coordinate = tile_coordinates[tile_index];
    float w = (coordinate.z + 0.5f) * w_step;

    for (int i = tid; i < (tile_size * tile_size); i += nr_threads)
    {
        int y = i / tile_size;
        int x = i % tile_size;

        if (y < tile_size) {
            // Compute phase
            const int x_ = (x + (tile_size / 2)) % tile_size;
            const int y_ = (y + (tile_size / 2)) % tile_size;

            // Use alternative computation of n to work around accuracy issues
            const float l = (x_ - (tile_size / 2)) * cell_size - shift[0];
            const float m = (y_ - (tile_size / 2)) * cell_size - shift[1];
            const float n = 1.0f - sqrtf(1.0 - (l * l) - (m * m));

            const float pi = (float) M_PI;
            const float phase = -2 * pi * n * w;

            // Compute phasor
            float2 phasor = make_float2(cosf(phase), sinf(phase)) * scale;

            // Apply correction
            size_t idx = index_grid(tile_size, tile_index, pol, y, x);
            tiles[idx] = tiles[idx] * phasor;
        }
    }
} // end kernel_apply_phasor

__global__ void kernel_subgrids_to_wtiles(
    const long                   grid_size,
    const int                    subgrid_size,
    const int                    tile_size,
    const int                    subgrid_offset,
    const Metadata* __restrict__ metadata,
    const float2*   __restrict__ subgrid,
          float2*   __restrict__ tiles,
          float2                 scale)
{
    // Map blockIdx.x to subgrids
    int s = blockIdx.x + subgrid_offset;

    // Map thread indices to thread id
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int tid = tidx + tidy * blockDim.x;

    // Compute the number of threads working on one subgrid
    int nr_threads = blockDim.x * blockDim.y;

    // Load tile coordinates
    const Metadata &m = metadata[s];
    int tile_index = m.wtile_index;
    int tile_top = m.wtile_coordinate.x * tile_size -
                    subgrid_size / 2 + grid_size / 2;
    int tile_left = m.wtile_coordinate.y * tile_size -
                    subgrid_size / 2 + grid_size / 2;

    // Compute position in tile
    int subgrid_x = m.coordinate.x - tile_top;
    int subgrid_y = m.coordinate.y - tile_left;

    // Iterate all pixels in subgrid
    for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads) {
        int y = i / subgrid_size;
        int x = i % subgrid_size;
        float pi = (float) M_PI;
        float phase = pi * (x+y-subgrid_size)/subgrid_size;
        float2 phasor = make_float2(cosf(phase) * scale.x,
                                    sinf(phase) * scale.y);

        if (y < subgrid_size)
        {
            // Compute shifted position in subgrid
            int x_src = (x + (subgrid_size/2)) % subgrid_size;
            int y_src = (y + (subgrid_size/2)) % subgrid_size;

            // Compute position in grid
            int x_dst = subgrid_x + x;
            int y_dst = subgrid_y + y;

            // Add subgrid value to grid
            #pragma unroll 4
            for (int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                long dst_idx = index_grid(tile_size + subgrid_size, tile_index, pol, y_dst, x_dst);
                long src_idx = index_subgrid(subgrid_size, s, pol, y_src, x_src);
                float2 value = phasor * subgrid[src_idx];
                atomicAdd(tiles[dst_idx], value);
            }
        }
    }
} // kernel_subgrids_to_wtiles

__global__ void kernel_wtiles_to_grid(
    const unsigned int             grid_size,
    const unsigned int             tile_size,
    const unsigned int             padded_tile_size,
    const int*        __restrict__ tile_ids,
    const Coordinate* __restrict__ tile_coordinates,
    const float2*   __restrict__   tiles,
          float2*   __restrict__   grid)
{
    // Map blockIdx.x to polarizations
    assert(gridDim.x == NR_POLARIZATIONS);
    unsigned int pol = blockIdx.x;

    // Map blockIdx.x to tiles
    unsigned int tile_index = tile_ids[blockIdx.y];

    // Map threadIdx.x to thread id
    unsigned int tid = threadIdx.x;

    // Compute the number of threads working on one polarizaton of a tile
    unsigned int nr_threads = blockDim.x;

    // Compute the padded size of the current tile
    const Coordinate& coordinate = tile_coordinates[blockIdx.y];

    // Compute position of tile in grid
    int x0 = coordinate.x * tile_size -
             (padded_tile_size - tile_size) / 2 + grid_size / 2;
    int y0 = coordinate.y * tile_size -
             (padded_tile_size - tile_size) / 2 + grid_size / 2;
    int x_start = max(0, x0);
    int y_start = max(0, y0);

    // Tranpose the polarizations
    const int index_pol_transposed[NR_POLARIZATIONS] = {0, 2, 1, 3};
    unsigned int pol_src = index_pol_transposed[pol];
    unsigned int pol_dst = pol;

    // Add tile to grid
    for (unsigned int i = tid; i < (padded_tile_size * padded_tile_size); i += nr_threads)
    {
        unsigned int y = i / padded_tile_size;
        unsigned int x = i % padded_tile_size;

        unsigned int y_dst = y_start + y;
        unsigned int x_dst = x_start + x;

        int y_src = y_dst - y0;
        int x_src = x_dst - x0;

        if (y < padded_tile_size)
        {
            unsigned long dst_idx = index_grid(grid_size, pol_dst, y_dst, x_dst);
            unsigned long src_idx = index_grid(padded_tile_size, tile_index, pol_src, y_src, x_src);
            atomicAdd(grid[dst_idx], tiles[src_idx]);
        }
    }
} // kernel_wtiles_to_grid

__global__ void kernel_subgrids_from_wtiles(
    const long                   grid_size,
    const int                    subgrid_size,
    const int                    tile_size,
    const int                    subgrid_offset,
    const Metadata* __restrict__ metadata,
          float2*   __restrict__ subgrid,
    const float2*   __restrict__ tiles,
          float2                 scale)
{
    // Map blockIdx.x to subgrids
    int s = blockIdx.x + subgrid_offset;

    // Map thread indices to thread id
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int tid = tidx + tidy * blockDim.x;

    // Compute the number of threads working on one subgrid
    int nr_threads = blockDim.x * blockDim.y;

    // Load tile coordinates
    const Metadata &m = metadata[s];
    int tile_index = m.wtile_index;
    int tile_top = m.wtile_coordinate.x * tile_size -
                    subgrid_size / 2 + grid_size / 2;
    int tile_left = m.wtile_coordinate.y * tile_size -
                    subgrid_size / 2 + grid_size / 2;

    // Compute position in tile
    int subgrid_x = m.coordinate.x - tile_top;
    int subgrid_y = m.coordinate.y - tile_left;

    // Iterate all pixels in subgrid
    for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads) {
        int y = i / subgrid_size;
        int x = i % subgrid_size;
        float pi = (float) M_PI;
        float phase = pi * (x+y-subgrid_size)/subgrid_size;
        float2 phasor = make_float2(cosf(phase) * scale.x,
                                    sinf(phase) * scale.y);

        if (y < subgrid_size)
        {
            // Compute shifted position in subgrid
            int x_src = (x + (subgrid_size/2)) % subgrid_size;
            int y_src = (y + (subgrid_size/2)) % subgrid_size;

            // Compute position in grid
            int x_dst = subgrid_x + x;
            int y_dst = subgrid_y + y;

            // Set subgrid value from grid
            #pragma unroll 4
            for (int pol = 0; pol < NR_POLARIZATIONS; pol++) {
                long src_idx = index_grid(tile_size + subgrid_size, tile_index, pol, y_dst, x_dst);
                long dst_idx = index_subgrid(subgrid_size, s, pol, y_src, x_src);
                subgrid[dst_idx] = tiles[src_idx] * phasor;
            }
        }
    }
} // kernel_subgrids_from_grid

__global__ void kernel_wtiles_from_grid(
    const unsigned int             dst_tile_size,
    const unsigned int             src_tile_size,
    const unsigned int             grid_size,
    const int*        __restrict__ tile_ids,
    const Coordinate* __restrict__ tile_coordinates,
          float2*     __restrict__ tiles,
    const float2*     __restrict__ grid)
{
    // Map blockIdx.x to polarizations
    assert(gridDim.x == NR_POLARIZATIONS);
    unsigned int pol = blockIdx.x;

    // Map blockIdx.x to tiles
    unsigned int tile_index = tile_ids[blockIdx.y];

    // Map threadIdx.x to thread id
    unsigned int tid = threadIdx.x;

    // Compute the number of threads working on one polarizaton of a tile
    unsigned int nr_threads = blockDim.x;

    // Compute the padded size of the current tile
    const Coordinate& coordinate = tile_coordinates[blockIdx.y];

    // Compute position of tile in grid
    int x0 = coordinate.x * dst_tile_size - (src_tile_size - dst_tile_size) / 2 +
             grid_size / 2;
    int y0 = coordinate.y * dst_tile_size - (src_tile_size - dst_tile_size) / 2 +
             grid_size / 2;
    int x_start = max(0, x0);
    int y_start = max(0, y0);

    // Tranpose the polarizations
    const int index_pol_transposed[NR_POLARIZATIONS] = {0, 2, 1, 3};
    unsigned int pol_src = pol;
    unsigned int pol_dst = index_pol_transposed[pol];

    // Add tile to grid
    for (unsigned int i = tid; i < (src_tile_size * src_tile_size); i += nr_threads)
    {
        unsigned int y = i / src_tile_size;
        unsigned int x = i % src_tile_size;

        unsigned int y_src = y_start + y;
        unsigned int x_src = x_start + x;

        unsigned int y_dst = y_src - y0;
        unsigned int x_dst = x_src - x0;

        if (y < src_tile_size)
        {
            unsigned long src_idx = index_grid(grid_size, pol_src, y_src, x_src);
            unsigned long dst_idx = index_grid(src_tile_size, tile_index, pol_dst, y_dst, x_dst);
            tiles[dst_idx] = grid[src_idx];
        }
    }
} // kernel_wtiles_from_grid

} // end extern "C"
