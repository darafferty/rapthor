#include "hip/hip_runtime.h"
#include "Types.h"
#include "math.cu"

#define ALIGN(N,A) (((N)+(A)-1)/(A)*(A))

__shared__ float4 shared[3][BATCH_SIZE];

/*
    Kernel
*/
extern "C" {
__global__ void
__launch_bounds__(BLOCK_SIZE)
kernel_degridder_1(
    const int                         grid_size,
    const int                         subgrid_size,
    const float                       image_size,
    const float                       w_step,
    const int                         nr_channels,
    const int                         nr_stations,
    const UVW*           __restrict__ uvw,
    const float*         __restrict__ wavenumbers,
          float2*        __restrict__ visibilities,
    const float*         __restrict__ spheroidal,
    const float2*        __restrict__ aterm,
    const Metadata*      __restrict__ metadata,
          float2*        __restrict__ subgrid)
{
    const unsigned UNROLL_TIME = 2;

    int s          = blockIdx.x;
    int tidx       = threadIdx.x;
    int tidy       = threadIdx.y;
    int tid        = tidx + tidy * blockDim.x;
    int nr_threads = blockDim.x * blockDim.y;

    // Load metadata for first subgrid
    const Metadata &m_0 = metadata[0];

    // Load metadata for current subgrid
    const Metadata &m = metadata[s];
    const int time_offset_global = (m.baseline_offset - m_0.baseline_offset) + m.time_offset;
    const int nr_timesteps = m.nr_timesteps;
    const int x_coordinate = m.coordinate.x;
    const int y_coordinate = m.coordinate.y;

    // Compute u,v,w offset in wavelenghts
    const float u_offset = (x_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
    const float v_offset = (y_coordinate + subgrid_size/2 - grid_size/2) / image_size * 2 * M_PI;
    const float w_offset = w_step * ((float) m.coordinate.z + 0.5) * 2 * M_PI;

    // Iterate visibilities
    for (int time = tid; time < ALIGN(nr_timesteps, nr_threads); time += nr_threads * UNROLL_TIME) {
        float2 visXX[UNROLL_TIME];
        float2 visXY[UNROLL_TIME];
        float2 visYX[UNROLL_TIME];
        float2 visYY[UNROLL_TIME];

        for (unsigned i = 0; i < UNROLL_TIME; i++) {
            visXX[i] = make_float2(0, 0);
            visXY[i] = make_float2(0, 0);
            visYX[i] = make_float2(0, 0);
            visYY[i] = make_float2(0, 0);
        }

        float u[UNROLL_TIME];
        float v[UNROLL_TIME];
        float w[UNROLL_TIME];

        for (unsigned i = 0; i < UNROLL_TIME; i++) {
            unsigned time_ = time + i * nr_threads;

            if (time_ < nr_timesteps) {
                u[i] = uvw[time_offset_global + time_].u;
                v[i] = uvw[time_offset_global + time_].v;
                w[i] = uvw[time_offset_global + time_].w;
            }
        }

        __syncthreads();

        // Iterate pixels
        const int nr_pixels = subgrid_size * subgrid_size;
        int current_nr_pixels = BATCH_SIZE;
        for (int pixel_offset = 0; pixel_offset < nr_pixels; pixel_offset += current_nr_pixels) {
            current_nr_pixels = nr_pixels - pixel_offset < min(nr_threads, BATCH_SIZE) ?
                                nr_pixels - pixel_offset : min(nr_threads, BATCH_SIZE);

            __syncthreads();

            // Prepare data
            for (int j = tid; j < current_nr_pixels; j += nr_threads) {
                int y = (pixel_offset + j) / subgrid_size;
                int x = (pixel_offset + j) % subgrid_size;

                // Compute shifted position in subgrid
                int x_src = (x + (subgrid_size/2)) % subgrid_size;
                int y_src = (y + (subgrid_size/2)) % subgrid_size;

                // Load pixels
                int idx_xx = index_subgrid(subgrid_size, s, 0, y_src, x_src);
                int idx_xy = index_subgrid(subgrid_size, s, 1, y_src, x_src);
                int idx_yx = index_subgrid(subgrid_size, s, 2, y_src, x_src);
                int idx_yy = index_subgrid(subgrid_size, s, 3, y_src, x_src);
                float2 pixelsXX = subgrid[idx_xx];
                float2 pixelsXY = subgrid[idx_xy];
                float2 pixelsYX = subgrid[idx_yx];
                float2 pixelsYY = subgrid[idx_yy];

                // Compute l,m,n and phase offset
                const float l = compute_l(x, subgrid_size, image_size);
                const float m = compute_m(y, subgrid_size, image_size);
                const float n = compute_n(l, m);
                float phase_offset = u_offset*l + v_offset*m + w_offset*n;

                // Store values in shared memory
                shared[0][j] = make_float4(pixelsXX.x, pixelsXX.y, pixelsXY.x, pixelsXY.y);
                shared[1][j] = make_float4(pixelsYX.x, pixelsYX.y, pixelsYY.x, pixelsYY.y);
                shared[2][j] = make_float4(l, m, n, phase_offset);
            } // end for j (pixels)

             __syncthreads();

            // Iterate current batch of pixels
            for (int k = 0; k < current_nr_pixels; k++) {
                // Load pixels from shared memory
                float2 apXX = make_float2(shared[0][k].x, shared[0][k].y);
                float2 apXY = make_float2(shared[0][k].z, shared[0][k].w);
                float2 apYX = make_float2(shared[1][k].x, shared[1][k].y);
                float2 apYY = make_float2(shared[1][k].z, shared[1][k].w);

                // Load l,m,n
                float l = shared[2][k].x;
                float m = shared[2][k].y;
                float n = shared[2][k].z;

                // Load phase offset
                float phase_offset = shared[2][k].w;

                // Load wavenumber
                float wavenumber = wavenumbers[0];

                // Iterate unrolled timesteps
                for (unsigned i = 0; i < UNROLL_TIME; i++) {
                    // Compute phase index
                    float phase_index = u[i] * l + v[i] * m + w[i] * n;

                    // Compute phasor
                    float  phase  = (phase_index * wavenumber) - phase_offset;
                    float2 phasor = make_float2(raw_cos(phase), raw_sin(phase));

                    // Multiply pixels by phasor
                    visXX[i].x += phasor.x * apXX.x;
                    visXX[i].y += phasor.x * apXX.y;
                    visXX[i].x -= phasor.y * apXX.y;
                    visXX[i].y += phasor.y * apXX.x;

                    visXY[i].x += phasor.x * apXY.x;
                    visXY[i].y += phasor.x * apXY.y;
                    visXY[i].x -= phasor.y * apXY.y;
                    visXY[i].y += phasor.y * apXY.x;

                    visYX[i].x += phasor.x * apYX.x;
                    visYX[i].y += phasor.x * apYX.y;
                    visYX[i].x -= phasor.y * apYX.y;
                    visYX[i].y += phasor.y * apYX.x;

                    visYY[i].x += phasor.x * apYY.x;
                    visYY[i].y += phasor.x * apYY.y;
                    visYY[i].x -= phasor.y * apYY.y;
                    visYY[i].y += phasor.y * apYY.x;
                }
            } // end for k (batch)
        } // end for j (pixels)

        for (unsigned i = 0; i < UNROLL_TIME; i++) {
            unsigned time_ = time + i * nr_threads;

            if (time_ < nr_timesteps) {
                // Store visibility
                const float scale = 1.0f / (nr_pixels);
                int idx_time = time_offset_global + time_;
                int idx_vis = index_visibility(1, idx_time, 0, 0);
                float4 visA = make_float4(visXX[i].x, visXX[i].y, visXY[i].x, visXY[i].y);
                float4 visB = make_float4(visYX[i].x, visYX[i].y, visYY[i].x, visYY[i].y);
                float4 *vis_ptr = (float4 *) &visibilities[idx_vis];
                vis_ptr[0] = visA * scale;
                vis_ptr[1] = visB * scale;
            }
        }
    } // end for time
} // end kernel_degridder_1
} // end extern "C"
