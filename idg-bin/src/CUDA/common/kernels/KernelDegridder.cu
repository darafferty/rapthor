#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>

#include "Types.h"
#include "math.cu"

#define MAX_NR_TIMESTEPS MAX_NR_TIMESTEPS_DEGRIDDER
#define NR_THREADS NR_THREADS_DEGRIDDER
#define ALIGN(N,A) (((N)+(A)-1)/(A)*(A))

/*
	Kernel
*/
template<int current_nr_channels> __device__ void kernel_degridder_(
    const float w_offset,
    const int nr_channels,
    const int channel_offset,
	const UVWType			__restrict__ uvw,
	const WavenumberType	__restrict__ wavenumbers,
	VisibilitiesType	    __restrict__ visibilities,
	const SpheroidalType	__restrict__ spheroidal,
	const ATermType			__restrict__ aterm,
	const MetadataType		__restrict__ metadata,
	const SubGridType	    __restrict__ subgrid
    ) {
    int tidx = threadIdx.x;
	int s = blockIdx.x;

    // Load metadata for first subgrid
    const Metadata &m_0 = metadata[0];

    // Load metadata for current subgrid
	const Metadata &m = metadata[s];
    const int time_offset_global = (m.baseline_offset - m_0.baseline_offset) + (m.time_offset - m_0.time_offset);
    const int nr_timesteps = m.nr_timesteps;
	const int aterm_index = m.aterm_index;
	const int station1 = m.baseline.station1;
	const int station2 = m.baseline.station2;
	const int x_coordinate = m.coordinate.x;
	const int y_coordinate = m.coordinate.y;

	// Compute u and v offset in wavelenghts
	float u_offset = (x_coordinate + SUBGRIDSIZE/2) / (float) IMAGESIZE;
	float v_offset = (y_coordinate + SUBGRIDSIZE/2) / (float) IMAGESIZE;

    // Shared data
    __shared__ float4 _pix[NR_POLARIZATIONS / 2][NR_THREADS];
	__shared__ float4 _lmn_phaseoffset[NR_THREADS];

    // Iterate all visibilities
    for (int i = tidx; i < ALIGN(nr_timesteps * current_nr_channels, NR_THREADS); i += NR_THREADS) {
        int time = i / current_nr_channels;
        int chan = i % current_nr_channels;

        float2 visXX, visXY, visYX, visYY;
        float  u, v, w;
        float  wavenumber;

        if (time < nr_timesteps) {
            visXX = make_float2(0, 0);
            visXY = make_float2(0, 0);
            visYX = make_float2(0, 0);
            visYY = make_float2(0, 0);

            u = uvw[time_offset_global + time].u;
            v = uvw[time_offset_global + time].v;
            w = uvw[time_offset_global + time].w;

            wavenumber = wavenumbers[chan + channel_offset];
        }

        for (int j = tidx; j < SUBGRIDSIZE * SUBGRIDSIZE; j += NR_THREADS) {
            int y = j / SUBGRIDSIZE;
            int x = j % SUBGRIDSIZE;

            __syncthreads();

            if (y < SUBGRIDSIZE) {
                float2 aXX1 = aterm[station1][aterm_index][0][y][x];
                float2 aXY1 = aterm[station1][aterm_index][1][y][x];
                float2 aYX1 = aterm[station1][aterm_index][2][y][x];
                float2 aYY1 = aterm[station1][aterm_index][3][y][x];

                // Load aterm for station2
                float2 aXX2 = hipConjf(aterm[station2][aterm_index][0][y][x]);
                float2 aXY2 = hipConjf(aterm[station2][aterm_index][1][y][x]);
                float2 aYX2 = hipConjf(aterm[station2][aterm_index][2][y][x]);
                float2 aYY2 = hipConjf(aterm[station2][aterm_index][3][y][x]);

                // Load spheroidal
                float _spheroidal = spheroidal[y][x];

                // Compute shifted position in subgrid
                int x_src = (x + (SUBGRIDSIZE/2)) % SUBGRIDSIZE;
                int y_src = (y + (SUBGRIDSIZE/2)) % SUBGRIDSIZE;

                // Load uv values
                float2 pixelsXX = _spheroidal * subgrid[s][0][y_src][x_src];
                float2 pixelsXY = _spheroidal * subgrid[s][1][y_src][x_src];
                float2 pixelsYX = _spheroidal * subgrid[s][2][y_src][x_src];
                float2 pixelsYY = _spheroidal * subgrid[s][3][y_src][x_src];

                // Apply aterm to subgrid
                float2 pixXX = pixelsXX * aXX1 + pixelsXY * aYX1 + pixelsXX * aXX2 + pixelsYX * aYX2;
                float2 pixXY = pixelsXX * aXY1 + pixelsXY * aYY1 + pixelsXY * aXX2 + pixelsYY * aYX2;
                float2 pixYX = pixelsYX * aXX1 + pixelsYY * aYX1 + pixelsXX * aXY2 + pixelsYX * aYY2;
                float2 pixYY = pixelsYX * aXY1 + pixelsYY * aYY1 + pixelsXY * aXY2 + pixelsYY * aYY2;

                // Store pixels
                _pix[0][tidx] = make_float4(pixXX.x, pixXX.y, pixXY.x, pixXY.y);
                _pix[1][tidx] = make_float4(pixYX.x, pixYX.y, pixYY.x, pixYY.y);

                float l = -(x - (SUBGRIDSIZE / 2)) * (float) IMAGESIZE / SUBGRIDSIZE;
                float m =  (y - (SUBGRIDSIZE / 2)) * (float) IMAGESIZE / SUBGRIDSIZE;
                float n = 1.0f - (float) sqrt(1.0 - (double) (l * l) - (double) (m * m));
                float phase_offset = u_offset * l + v_offset * m + w_offset * n;
                _lmn_phaseoffset[tidx] = make_float4(l, m, n, phase_offset);
            }

            __syncthreads();

            if (time < nr_timesteps) {
                #if SUBGRIDSIZE * SUBGRIDSIZE % NR_THREADS == 0
                int last_k = NR_THREADS;
                #else
                int first_j = j / NR_THREADS * NR_THREADS;
                int last_k =  first_j + NR_THREADS < SUBGRIDSIZE * SUBGRIDSIZE ? NR_THREADS : SUBGRIDSIZE * SUBGRIDSIZE - first_j;
                #endif

                for (int k = 0; k < last_k; k ++) {
                    // Load l,m,n
                    float  l = _lmn_phaseoffset[k].x;
                    float  m = _lmn_phaseoffset[k].y;
                    float  n = _lmn_phaseoffset[k].z;

                    // Load phase offset
                    float phase_offset = _lmn_phaseoffset[k].w;

                    // Compute phase index
                    float phase_index = u * l + v * m + w * n;

                    // Compute phasor
                    float  phase  = (phase_index * wavenumber) - phase_offset;
                    float2 phasor = make_float2(cosf(phase), sinf(phase));

                    // Load pixels from shared memory
                    float2 apXX = make_float2(_pix[0][k].x, _pix[0][k].y);
                    float2 apXY = make_float2(_pix[0][k].z, _pix[0][k].w);
                    float2 apYX = make_float2(_pix[1][k].x, _pix[1][k].y);
                    float2 apYY = make_float2(_pix[1][k].z, _pix[1][k].w);

                    // Multiply pixels by phasor
                    visXX.x += apXX.x * phasor.x;
                    visXX.x -= apXX.y * phasor.y;
                    visXX.y += apXX.x * phasor.y;
                    visXX.y += apXX.y * phasor.x;

                    visXY.x += apXY.x * phasor.x;
                    visXY.x -= apXY.y * phasor.y;
                    visXY.y += apXY.x * phasor.y;
                    visXY.y += apXY.y * phasor.x;

                    visYX.x += apYX.x * phasor.x;
                    visYX.x -= apYX.y * phasor.y;
                    visYX.y += apYX.x * phasor.y;
                    visYX.y += apYX.y * phasor.x;

                    visYY.x += apYY.x * phasor.x;
                    visYY.x -= apYY.y * phasor.y;
                    visYY.y += apYY.x * phasor.y;
                    visYY.y += apYY.y * phasor.x;
                }
            }
        }

        // Set visibility value
        int vis_offset = ((time_offset_global + time) * nr_channels) + channel_offset;
        if (time < nr_timesteps) {
            visibilities[vis_offset + chan][0] = visXX;
            visibilities[vis_offset + chan][1] = visXY;
            visibilities[vis_offset + chan][2] = visYX;
            visibilities[vis_offset + chan][3] = visYY;
        }
    }
}

extern "C" {
__global__ void kernel_degridder(
    const float w_offset,
    const int nr_channels,
	const UVWType			__restrict__ uvw,
	const WavenumberType	__restrict__ wavenumbers,
	VisibilitiesType	    __restrict__ visibilities,
	const SpheroidalType	__restrict__ spheroidal,
	const ATermType			__restrict__ aterm,
	const MetadataType		__restrict__ metadata,
	const SubGridType	    __restrict__ subgrid
	) {
    int channel_offset = 0;
    for (; (channel_offset + 8) <= nr_channels; channel_offset += 8) {
        kernel_degridder_<8>(
            w_offset, nr_channels, channel_offset, uvw, wavenumbers,
            visibilities,spheroidal, aterm, metadata, subgrid);
    }

    for (; channel_offset < nr_channels; channel_offset++) {
        kernel_degridder_<1>(
            w_offset, nr_channels, channel_offset, uvw, wavenumbers,
            visibilities,spheroidal, aterm, metadata, subgrid);
    }
}
}
